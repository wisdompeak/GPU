#include "hip/hip_runtime.h"
#include <cmath>
#include <fstream>
#include <time.h>
#include <iostream>
#include <stddef.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>

#define GPU_Device 2

using namespace std;
        
// Mode selection
const int FISTA = 0;
    // -2: execution configuration test
    // -1: joint operator test
    // 0: SART + denoise
    // 1: SART + FISTA

const int DenoiseOption = 1;

const float Lip_con = 16.0f;          //
const float lambda_TV = 10.0;         //regularization parameter for the tv norm
const float lambda_L1 = 0.00f;         //regularization parameter for the l1 norm

// Distances
const float DSO = 0.825f;
const float DOD = -0.625f;

/* Parameters for reconstructed volume
 * You need to consider the GPU memory capcity
 * Double check the GPU block configuration
 */
const int M = 256;      // reconstruction volume x range       maximum: 1024
const int N = 256;      // reconstruction volume y range       maximum: 1024
const int ZETA = 128;   // reconstruction volume z range       maximum: 1024 

const int NO_X = M;
const int NO_Y = N;
const int NO_Z = ZETA;
const int NO_VOXEL = NO_X*NO_Y*NO_Z; 

const float volumn_x = 1.5e-3 ; // volume voxel physical size (in m)
const float inv_volumn_x = 1.0/volumn_x; 
const int M_Offset = 0;
const float boundary_voxel_x  = -volumn_x*(float(M)/2.0f+M_Offset);

const float volumn_y = volumn_x ;
const float inv_volumn_y = 1.0/volumn_y; 
const float boundary_voxel_y  = -volumn_y*(float(N)/2.0f);

const float volumn_z = volumn_x ;
const float inv_volumn_z = 1.0/volumn_z; 
const float boundary_voxel_z = -volumn_z*(float(ZETA)/2.0f); 

// parameters for half detector offset
const float Offset =  0;

// parameters for Detector
const int R = 275;         // detector width (in pixel)
const int Z_prj = 5;       // detector height (in pixel)
const float Detector_pixel_x = 2.54e-3; // all in m
const float Detector_Ymin = -Detector_pixel_x*(float(R)/2.0f - 0.5f) + Offset; 
const float Detector_Zmin = -Detector_pixel_x*(float(Z_prj)/2.0f - 0.5f); 
const float Detector_Zmax = +Detector_pixel_x*(float(Z_prj)/2.0f - 0.5f); 

const float PI = 3.141592653589793f; 

// parameters for Source  
const int N_source = 75;    // number of sources
const float Source_interval = 4e-3;   // source spacing
const float Source_z_min = -Source_interval*(float(N_source)/2.0f - 0.5f);
const float Source_y = Offset;
const float Source_x = DSO;

// parameters for Acqusition
const int Nviews = 360; 
const float us_rate = 1.00f; 
const float initialAngle= 0.00f ;
const float shiftAngle= 0.0f;

const float MAX_infi = 1e16;

#include "InitGPU.h"
#include "kernel_tool_functions.cu"
#include "host_tool_functions.cu"
// #include "kernel_otherFunctions.cu"


/* If you want to use the classical version in which the backprojection is implemented in voxel-driven, 
 * please uncomment the follwing two files and comment out the corresponding new versions.
 */
#include "voxel/kernel_IterativeRecon_TBCT.cu"
#include "voxel/host_IterativeRecon_TBCT.c"

// #include "kernel_IterativeRecon_TBCT_new.cu"
// #include "host_IterativeRecon_TBCT_new.c"

#include "host_FGP_Denoise_CPU.h"

#include "kernel_FGP_Denoise_GPUx4.cu"
#include "host_FPG_Denoise_GPUx4.c"

#include "kernel_FGP_Denoise_GPUx7.cu"
#include "host_FGP_Denoise_GPUx7.c"


main(int argc, char ** argv)
{
    
    // print CUDA information
    if (!InitCUDA()) 
    {
        return 0;
    }    
    
    /* ************* User defined parameters ************/
    
    char directory[]="/home/richard/TBCT/PigHead/";
    char objectName[]="PigHead";
    char outputFolder[]="/Reconstructed_images_TBCT/";   
    
    int Iter_num = 31; 
    
	int Niter_denoise = 20;         //iter number for denoising problem

    /*********** other declared variables ************/
    
    float step_size = 2.0f/Lip_con;
    float lambda_denoise_TV = 2.0f*lambda_TV/Lip_con;
           
	double data_fidelity;
	double tv_value = 0.0f;
	double object_function_value_xk;       
    double *object_function_array = new double [Iter_num*3];
	bzero(object_function_array, sizeof(double)*Iter_num*3);  

    FILE *fp;		    
    char filename[200];
    char fn[200];
    int VIEW = Nviews;    
    float endAngle = initialAngle + (VIEW - 1)*us_rate;  
    float fileAngle;
    // variable definition: G = H*F  
    // G: measured projection; H: system matrix; F: object (reconstruction volume)
    
    // for 3D reconstructed volume
	float *F_Y_k = new float [M*N*ZETA];    // Y(k)
	bzero(F_Y_k, sizeof(float)*M*N*ZETA);

    float *F_X_k_1 = new float [M*N*ZETA];  // X(k-1)
    bzero(F_X_k_1, sizeof(float)*M*N*ZETA);
    
    float *F_recon = new float [M*N*ZETA];  // X(k-1)
    bzero(F_recon, sizeof(float)*M*N*ZETA);    
    
    // for 2D projection data set
	float *h_proj_measured = new float [R*Z_prj*N_source*Nviews];
	bzero(h_proj_measured, sizeof(float)*R*Z_prj*N_source*Nviews);
  
    
	/********** Read Projections **************/

    printf("Read projection files ...\n");
    
	for (int j=0;j<Nviews;j++)
	{
        fileAngle = float(j*us_rate + initialAngle);        
    
        strcpy(filename,directory);
 		sprintf(fn,"/FinalData/proj_%.2f.bin", fileAngle);
        strcat(filename,fn);
        //cout<<fn<<endl;
        if ( (fp = fopen(filename,"rb")) == NULL )
        {
        	printf("Can not open projection files for main function \n");
            printf("%s\n",filename);                    
            exit(0);
        }
//         fseek(fp,sizeof(float)*R*(int(2048/2-Z_prj/2)),0); // If you want to read part of the projections
        fread(h_proj_measured + j*R*Z_prj*N_source, sizeof(float)*R*Z_prj*N_source,1,fp);  // stack all projections together
        fclose(fp);
	}       
    
    
    /***********************************************************************************/
    
	size_t size_proj_data = sizeof(float)*Z_prj*R*N_source*Nviews;
	size_t size_volume = sizeof(float)*M*N*ZETA;
    
    // allocate GPU memory for the whole measurement data
	float *d_proj_data;    
	hipMalloc((void**)&d_proj_data, size_proj_data);
    hipMemcpy(d_proj_data, h_proj_measured, size_proj_data, hipMemcpyHostToDevice);  // load data into GPU
    
    // allocate GPU memory for the recon volume
	float *d_recon = NULL;
	hipMalloc((void**)&d_recon, size_volume);    
    hipMemset(d_recon, 0, size_volume); 
    
    /**************** Inverse Crime Studies ********************************/
    
    // load volumetric image
    
//         strcpy(filename,directory);
//  		sprintf(fn,"SLphantom3d_256.dat");        
//         strcat(filename,fn);
//         cout<<"Loading "<<fn<<endl;
//         if ( (fp = fopen(filename,"rb")) == NULL )
//                 {
//                     printf("Can not load volumetric image \n");
//                     printf("%s\n",filename);                    
//                     exit(0);
//                 }
//         fread(F_recon, sizeof(float)*M*N*ZETA,1,fp);
//         fclose(fp);             
//         cout<<"Load Phantom Sucessfully!"<<endl;
            
//         Forward_3D_ray_driven_siddon_TBCT(F_recon,h_proj_measured);  
//     
//         if ( (fp = fopen("../GeneratedProjection.dat","wb")) == NULL )
//         	{
//                 	printf("can not open file to write the generated proj data \n");
//                     printf("%s\n",filename);
//                 	exit(0);
//         	}
//         fwrite(h_proj_measured,sizeof(float)*R*Z_prj*N_source*Nviews,1,fp);
//         fclose(fp);     
//     
// //         bzero(F_recon,sizeof(float)*M*N*ZETA);
        
	/****************Iteration Reconstruction******************************/
	

    float t_k;
    float t_k_1=1.0f;    
            
    
    //Set Timer 1
    struct timeval t1,t2;
    gettimeofday(&t1,NULL);    
    
	for (int k=1;k<=Iter_num;k++)
	{	
//         if (FISTA==-2)  // "occupancy calculator", check the best execution configuration. Refer to the program guide
//         {
//             int numBlocks;       // Occupancy in terms of active blocks
//             int blockSize = 256;
//             int activeWarps;
//             int maxWarps;
// 
//             hipDeviceProp_t prop;
//             hipGetDeviceProperties(&prop, GPU_Device);
//             
//             hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks,backprj_ray_driven_3d_kernel,blockSize,0);
//             activeWarps = numBlocks * blockSize / prop.warpSize;
//             maxWarps = prop.maxThreadsPerMultiProcessor / prop.warpSize;
//             std::cout << "Occupancy: " << (double)activeWarps / maxWarps * 100 << "%" << std::endl;      
//             std::cout << "MaxActiveBlocksPerMultiprocessor: " << numBlocks << std::endl;      
//             goto endProgram;   
//         }
        
        if (FISTA==-1)  // check matched joint operator;
        {
            /*Note: You need to first uncomment the phantom loading code to initialize a valid F_recon*/            
            CheckMatchedJointOperator(F_recon);  
            goto endProgram;   
        }
                
        if (FISTA==0)
        {
        
            printf("Undergoing SART updating...  relaxation = %f\n", step_size);			
                        
            Reconstruction_3D_ray_driven_TBCT(d_recon, d_proj_data, step_size);           
    
//             SaveDeviceDataToFile(d_recon,M*N*ZETA,"../Recon.dat");            
    
            if (lambda_TV>0.0f)                    
            {
                printf("Undergoing TV regularization ...\n");                                
                switch(DenoiseOption) // Denoise options
                {
                    case 1 : FGP_denoise_GPUx7_exact(d_recon, lambda_denoise_TV, Niter_denoise); 
                             break;     // require x7 volume size, fast
                    case 2 : FGP_denoise_GPUx4_exact(d_recon, lambda_denoise_TV, Niter_denoise); 
                             break;     // require x4 volume size, slowest
                    case 3 : FGP_denoise_GPUx4_apprx(d_recon, lambda_denoise_TV, Niter_denoise); 
                             break;     // require x4 volume size, slower
                    case 4 : GP_denoise_GPUx4_fast(d_recon, lambda_denoise_TV, Niter_denoise); 
                             break;     // require x4 volume size, fast, slow in convergence
                }
                std::cout<<" - TV regularization finished.\n";
            }
            
            hipMemcpy(F_recon, d_recon, size_volume, hipMemcpyDeviceToHost);
        }
        
        
        if (FISTA==1)
        {
            printf("Undergoing SART updating...  relaxation = %f\n", step_size);					            
        
            hipMemcpy(d_recon, F_Y_k, size_volume, hipMemcpyHostToDevice); 
            
            Reconstruction_3D_ray_driven_TBCT(d_recon, d_proj_data, step_size);          

            if (lambda_TV>0.0f)                    
            {
                printf("Undergoing TV regularization ...\n");          
                switch(DenoiseOption) // Denoise options
                {
                    case 1 : FGP_denoise_GPUx7_exact(d_recon, lambda_denoise_TV, Niter_denoise); 
                             break;     // require x7 volume size, fast
                    case 2 : FGP_denoise_GPUx4_exact(d_recon, lambda_denoise_TV, Niter_denoise); 
                             break;     // require x4 volume size, slowest
                    case 3 : FGP_denoise_GPUx4_apprx(d_recon, lambda_denoise_TV, Niter_denoise); 
                             break;     // require x4 volume size, slower
                    case 4 : GP_denoise_GPUx4_fast(d_recon, lambda_denoise_TV, Niter_denoise); 
                             break;     // require x4 volume size, fast, slow in convergence
                }
                std::cout<<" - TV regularization finished.\n";
            }           
            
            hipMemcpy(F_recon, d_recon, size_volume, hipMemcpyDeviceToHost);   
            
            t_k = (1.0f + sqrt(1.0f + 4.0f*t_k_1*t_k_1) )/2.0f;        
                        // Note: t(k) = [1+sqrt(1+4*t(k-1)^2)]/2
            for (int i=0;i<NO_VOXEL;i++)
            	F_Y_k[i] = F_recon[i] + (t_k_1 -1.0f)/t_k * (F_recon[i] - F_X_k_1[i]);
                        // Note: y(k) = x(k) + [t(k-1) -1]/t(k) * [x(k)-x(k-1)]
            t_k_1 = t_k;
                        // Note: Update t(k-1):   t(k-1) <- t(k)
            memcpy(F_X_k_1,F_recon,sizeof(float)*M*N*ZETA);
                        // Note: Update x(k-1):   x(k-1) <- x(k)
            
        }
        
            /*****************Calculating Obj Func Value ********************/

            std::cout<<"Calculating Object Func Value ...\n";            
                //Note: object function value || Ax - b ||_2 + 2*lambda_TV*||f||_tvnorm  + lambda_L1*||\phi f ||_L1 ;
                        
            /*** data fidelity ****/
			std::cout<<" - calculating data fidelity ... \n";	
            
            float *d_proj_forward = NULL;
            hipMalloc((void**)&d_proj_forward, size_proj_data);    
            hipMemset(d_proj_forward, 0, size_proj_data);             
            Forward_3D_ray_driven_siddon_TBCT(d_recon, d_proj_forward);
                        
            data_fidelity = L2_norm_gpu(d_proj_forward, d_proj_data);
           	std::cout<<"    * L2 Norm="<<data_fidelity<<endl;              
            hipFree(d_proj_forward);
            
            /*** TV norm ****/
            std::cout<<" - calculating TV norm ... \n";
            tv_value = TV_norm_gpu(d_recon);                        
            std::cout<<"    * TV value="<<tv_value<<endl;  
            

            /***** obj function ******/
            object_function_value_xk = data_fidelity + 2.0f*lambda_TV*tv_value;
                //Note: object_function_value_xk = data_fidelity + 2.0f*lambda_TV*tv_value + 1.0f*lambda_L1*l1_value;
            object_function_array[k*3-3] = tv_value;
            object_function_array[k*3-2] = data_fidelity;
            object_function_array[k*3-1] = object_function_value_xk;
           	std::cout<<"Object function value for x(k) = "<< tv_value <<" + "<< data_fidelity <<" = "<<object_function_value_xk <<std::endl; 
                   

            /***************** Saving ********************/            
            
            strcpy(filename,directory);
            sprintf(fn,"%s/%s_%d_%d_%d_%.0fum_new_view_%d_(%.0f,%.0f)_TV_%.2f_L1_%.2f_Lip_%.2f.recon",outputFolder, objectName, M,N,ZETA, volumn_x*1000000, Nviews, initialAngle, endAngle, lambda_TV, lambda_L1, Lip_con);
            strcat(filename,fn);        	
        	if ( (fp = fopen(filename,"wb")) == NULL )
        	{
                	printf("can not open file to write the intermediate reconstructed image \n");
                    printf("%s\n",filename);
                	exit(0);
        	}
        	fwrite(F_recon,sizeof(float)*M*N*ZETA,1,fp);
        	fclose(fp);  
            
            strcpy(filename,directory);
            sprintf(fn,"%s/object_func_%s_view_%d_(%.0f,%.0f)_TV_%.2f_Lip_%.2f.bin",outputFolder, objectName, Nviews, initialAngle, endAngle, lambda_TV, Lip_con);
            strcat(filename,fn);                 
        	if ( (fp = fopen(filename,"wb")) == NULL )
        	{
                	printf("can not open file to write the tv_value_file \n");
                    printf("%s\n",filename);
                	exit(0);
        	}
        	fwrite(object_function_array,sizeof(double)*k*3,1,fp);
        	fclose(fp);
		
            if (k%10==1)
            {
                strcpy(filename,directory);
                sprintf(fn,"%s/%s_%d_%d_%d_%.0fum_iterative_%d_view_%d_(%.0f,%.0f)_TV_%.2f_L1_%.2f_Lip_%.2f.recon",outputFolder, objectName, M,N,ZETA, volumn_x*1000000, k, Nviews, initialAngle, endAngle, lambda_TV, lambda_L1, Lip_con);
                strcat(filename,fn);                
        		if ( (fp = fopen(filename,"wb")) == NULL )
        		{
                		printf("can not open file to write the reconstructed image \n");
                        printf("%s\n",filename);
                		exit(0);
        		}
        		fwrite(F_recon,sizeof(float)*M*N*ZETA,1,fp);
        		fclose(fp);
            }

            std::cout<<"Have done "<< k <<" iteration(s)"<<std::endl<<endl;
	}

    gettimeofday(&t2,NULL);
    printf("Whole computing (gettimeofday): %f (s)\n\n\n", (t2.tv_sec-t1.tv_sec + (t2.tv_usec-t1.tv_usec)/1000000.0));

	endProgram: ;

    hipFree(d_proj_data);
    hipFree(d_recon);
    
	delete []F_Y_k;
    delete []F_X_k_1;
    delete []F_recon;    
    delete []h_proj_measured;
    delete []object_function_array;

	return 0;
}
