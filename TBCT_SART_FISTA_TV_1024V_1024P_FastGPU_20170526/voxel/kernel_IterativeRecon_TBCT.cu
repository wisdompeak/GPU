#include "hip/hip_runtime.h"
__global__ void forward_ray_driven_3d_kernel_correction(float *d_f , float *d_proj_correction, float *d_proj_data, float sin_theta, float cos_theta, int command)
{
    /* 
     * Reference: 
     * "A fast algorithm to calculate the exact radiological path through a pixel or voxel space"
     * Filip Jacobs,
     * 
     * "Accelerating simultaneous algebraic reconstruction technique with motion compensation using CUDA-enabled GPU" 
     * Wai-Man Pang, CUHK
     * Section: Back-projection and image update
     */
    
	// d_f: 3D object array;    d_f[i,j,k] = d_f [k*M*N+j*M+i]; 
    // d_proj_correction: weighted 2D projection correction,  (output of this function. i.e. c(i) in the paper)    
    // d_proj_data: pointer of the 2D projection acquired at the angle of t_theta	
    
    /* Note:
     * dim3  dimGrid(Z_prj,N_source);  
     * dim3  dimBlock(R,1); 
     */
    
    int proj_x_idx = threadIdx.x;
    int proj_z_idx = blockIdx.x;
    int proj_src_idx = blockIdx.y;     
    
    // X2 point coordinate in (x,y,z) system . the source position
    float vertex_x2_x, vertex_x2_y, vertex_x2_z;
    vertex_x2_x = Source_x * cos_theta - Source_y * sin_theta;
    vertex_x2_y = Source_x * sin_theta + Source_y * cos_theta;
    vertex_x2_z = Source_z_min + proj_src_idx * Source_interval;

    //X1 point coordinate in (x,y,z) system --- detector pixel positions 
    float vertex_x1_x, vertex_x1_y, vertex_x1_z;
    vertex_x1_x = DOD * cos_theta - (Detector_Ymin +  proj_x_idx * Detector_pixel_x) * sin_theta;
    vertex_x1_y = DOD * sin_theta + (Detector_Ymin +  proj_x_idx * Detector_pixel_x) * cos_theta;
    vertex_x1_z = (Detector_Zmin + proj_z_idx * Detector_pixel_x);        
        //  Notice: vertex_x1_x < 0 < vertex_x2_x    
    
    int proj_pixel_index = R * Z_prj * proj_src_idx + R * proj_z_idx + proj_x_idx;       
    
    /***************/
                
    float alpha_x_0= 0.0f, alpha_x_1= 0.0f, alpha_x_M= 0.0f;
    float alpha_y_0= 0.0f, alpha_y_1= 0.0f, alpha_y_N= 0.0f;
    float alpha_z_0= 0.0f, alpha_z_1= 0.0f, alpha_z_T= 0.0f;
    float alpha_x_min= 0.0f, alpha_y_min= 0.0f, alpha_z_min= 0.0f;
    float alpha_x_max= 0.0f, alpha_y_max= 0.0f, alpha_z_max= 0.0f;
    float alpha_min= 0.0f, alpha_max= 0.0f;
        
	int i_min=0, j_min=0, k_min=0;
	int i_max=0, j_max=0, k_max=0;
    int i=0, j=0, k=0;
    int voxel_i=0, voxel_j=0, voxel_k=0;
    float Error=0.0f;
    
	float alpha_x=0.0f, alpha_y=0.0f, alpha_z=0.0f;  
    float one_ray_sum = 0.0f;
    float one_ray_length = 0.0f; 

    float alpha_c= 0.0f;
    float d_x1_x2= 0.0f;

	int N_total_sec=0; 
        
	/**** Step 1 :find out alpha_min, alpha_max ********/

    if ( (vertex_x1_x == vertex_x2_x) || (vertex_x1_y == vertex_x2_y)  )  //Note: You may rotate the angle to avoid this happening
    {    
        d_proj_correction[proj_pixel_index] = 0.0f ;
        Error = 1.0f;
    }    

    else if ( (vertex_x1_x != vertex_x2_x) && (vertex_x1_y != vertex_x2_y)  )
    {

        alpha_x_0 = (boundary_voxel_x + volumn_x*0 - vertex_x1_x )/ (vertex_x2_x - vertex_x1_x); //(9)
        alpha_x_M = (boundary_voxel_x + volumn_x*M - vertex_x1_x )/ (vertex_x2_x - vertex_x1_x);
        // Notice: it is still unsure here which one is the parametric value of the first intersection point of the ray with the x-plane
        // It depends on whether source or detector lies on the left side of the reconstruction region at this time
    
        alpha_y_0 = (boundary_voxel_y + volumn_y*0 - vertex_x1_y )/ (vertex_x2_y - vertex_x1_y);
        alpha_y_N = (boundary_voxel_y + volumn_y*N - vertex_x1_y )/ (vertex_x2_y - vertex_x1_y);
        
        if (fabs(vertex_x2_z - vertex_x1_z) < volumn_z*1e-6)    // in case x1 and x2 are at the same z position
        {
            alpha_z_0 = -MAX_infi;
            alpha_z_T = MAX_infi;
            Error = 0.0;
        }
        else
        {
            alpha_z_0 = (boundary_voxel_z + volumn_z*0 - vertex_x1_z )/ (vertex_x2_z - vertex_x1_z);
            alpha_z_T = (boundary_voxel_z + volumn_z*ZETA - vertex_x1_z )/ (vertex_x2_z - vertex_x1_z);
            Error = 0.0;            
        }
       
        alpha_x_min = fmin(alpha_x_0, alpha_x_M);   //(5)
        alpha_x_max = fmax(alpha_x_0,  alpha_x_M );  //(6)
        
        alpha_y_min = fmin(alpha_y_0, alpha_y_N);   //(7)
        alpha_y_max = fmax(alpha_y_0,  alpha_y_N );  //(8)
        
        alpha_z_min = fmin(alpha_z_0, alpha_z_T);               
        alpha_z_max = fmax(alpha_z_0,  alpha_z_T );
        
        alpha_min = (fmax(fmax(alpha_x_min, alpha_y_min), fmax(alpha_y_min, alpha_z_min))); //(3)
        // i.e. alpha_min = fmax(alpha_x_min,alpha_y_min,alpha_z_min)
        // it indicates the point where the path interacts with the near boundary of reconstruction region        

        alpha_max = (fmin(fmin(alpha_x_max, alpha_y_max), fmin(alpha_y_max, alpha_z_max))); //(4)
        // i.e. alpha_max = fmin(alpha_x_max,alpha_y_max,alpha_z_max)
        // it indicates the point where the path last interacts with the far boundary of reconstruction region        
        
        /********Step 2,3: Find i_max, i_min***************/
        
     if (alpha_max <= alpha_min)
     {
                d_proj_correction[proj_pixel_index] = 0.0f ;
     }
  
	 else if ( (alpha_max > alpha_min) && (alpha_min > 0.0f)  )
     {
			// X direction 
			if (vertex_x1_x < vertex_x2_x)
			{	
				if (alpha_min == alpha_x_min)
					i_min = 1;      //(11)
				else if (alpha_min != alpha_x_min)
					i_min =  floor(( alpha_min*(vertex_x2_x - vertex_x1_x) + vertex_x1_x  - boundary_voxel_x )*inv_volumn_x) + 1 ;
                                    //(12)
                     /* Note: i_min is the index of the 1st x plane where the path interacts inside the reconstruction region
                      * It is not the index of alpha_x_min
                      */                
				if (alpha_max == alpha_x_max)
					i_max = M;      //(13)
				else if (alpha_max != alpha_x_max)
					i_max =  floor(( alpha_max*(vertex_x2_x - vertex_x1_x) + vertex_x1_x  - boundary_voxel_x )*inv_volumn_x) ;
                                    //(14)
                     // Note: i_max is the index of the last x plane where the path interacts with the reconstruction region (inside or boundary)                      
			}	
			else if (vertex_x1_x > vertex_x2_x)
			{	
				if (alpha_min == alpha_x_min)
					i_max = M-1;    //(15)
				else if (alpha_min != alpha_x_min)
					i_max =  floor(( alpha_min*(vertex_x2_x - vertex_x1_x) + vertex_x1_x  - boundary_voxel_x )*inv_volumn_x) ;				
                                    //(16)
				if (alpha_max == alpha_x_max)
					i_min = 0;      //(17)
				else if (alpha_max != alpha_x_max)
					i_min =  floor(( alpha_max*(vertex_x2_x - vertex_x1_x) + vertex_x1_x  - boundary_voxel_x )*inv_volumn_x) + 1 ;
                                    //(18)
			}	
            // Note: overall, i_min is the most left x-plane, i_max the most right x-plane,
            // and the initial point (the first interacted position on the boundary) NOT included.            
               
			//Y direction 
			if (vertex_x1_y < vertex_x2_y)
			{	
				if (alpha_min == alpha_y_min)
					j_min = 1; 
				else if (alpha_min != alpha_y_min)
					j_min =  floor(( alpha_min*(vertex_x2_y - vertex_x1_y) + vertex_x1_y  - boundary_voxel_y )*inv_volumn_y) + 1 ;
				
				if (alpha_max == alpha_y_max)
					j_max = N; 
				else if (alpha_max != alpha_y_max)
					j_max =  floor(( alpha_max*(vertex_x2_y - vertex_x1_y) + vertex_x1_y  - boundary_voxel_y )*inv_volumn_y) ;

			}	
			else if (vertex_x1_y > vertex_x2_y)
			{	
				if (alpha_min == alpha_y_min)
					j_max = N-1; 
				else if (alpha_min != alpha_y_min)
					j_max =  floor(( alpha_min*(vertex_x2_y - vertex_x1_y) + vertex_x1_y  - boundary_voxel_y )*inv_volumn_y) ;
				
				if (alpha_max == alpha_y_max)
					j_min = 0; 
				else if (alpha_max != alpha_y_max)
					j_min =  floor(( alpha_max*(vertex_x2_y - vertex_x1_y) + vertex_x1_y  - boundary_voxel_y )*inv_volumn_y) + 1 ;

			}	
            // Note: overall, j_min is the most bottom y-plane, j_max the most top y-plane,
            // and the initial point (the first interacted position on the boundary) NOT included.
            
			//Z direction 
            if (fabs(vertex_x1_z-vertex_x2_z)<volumn_z*1e-6 )  
            {
					k_min =  floor(( vertex_x1_z  - boundary_voxel_z )*inv_volumn_z) + 1 ;                
					k_max =  floor(( vertex_x1_z  - boundary_voxel_z )*inv_volumn_z) ;    
            }
            else if (vertex_x1_z < vertex_x2_z)
			{	
				if (alpha_min == alpha_z_min)
					k_min = 1; 
				else if (alpha_min != alpha_z_min)
					k_min =  floor(( alpha_min*(vertex_x2_z - vertex_x1_z) + vertex_x1_z  - boundary_voxel_z )*inv_volumn_z) + 1 ;
				
				if (alpha_max == alpha_z_max)
					k_max = ZETA; 
				else if (alpha_max != alpha_z_max)
					k_max =  floor(( alpha_max*(vertex_x2_z - vertex_x1_z) + vertex_x1_z  - boundary_voxel_z )*inv_volumn_z) ;

			}	
			else if (vertex_x1_z > vertex_x2_z)
			{	
				if (alpha_min == alpha_z_min)
					k_max = ZETA-1; 
				else if (alpha_min != alpha_z_min)
					k_max =  floor(( alpha_min*(vertex_x2_z - vertex_x1_z) + vertex_x1_z  - boundary_voxel_z )*inv_volumn_z) ;
				
				if (alpha_max == alpha_z_max)
					k_min = 0; 
				else if (alpha_max != alpha_z_max)
					k_min =  floor(( alpha_max*(vertex_x2_z - vertex_x1_z) + vertex_x1_z  -boundary_voxel_z )*inv_volumn_z) + 1 ;

			}	
            
        /************ initialization (i,j,k) (alpha_x_1,y,z)**************************/            
        // Note: (i,j,k) is the current x,y,z plane index (@ the initial point at the boundary)
        // Note: (alpha_x_1,alpha_y_1,alpha_z_1) is the next x,y,z plane to go.
            
		N_total_sec = i_max - i_min + 1 + j_max - j_min +1 + k_max-k_min +1; 
            // i.e. N_p (25)

		if (vertex_x1_x < vertex_x2_x)
        {
 			alpha_x_1 = (volumn_x * (float)i_min  +boundary_voxel_x - vertex_x1_x )/ (vertex_x2_x - vertex_x1_x);
            i = i_min - 1;   
        }        
		else if (vertex_x1_x > vertex_x2_x) 			
        {
            alpha_x_1 = (volumn_x * (float)i_max  +boundary_voxel_x - vertex_x1_x )/ (vertex_x2_x - vertex_x1_x);
            i = i_max + 1;
        }
            // Note: alpha_x_1 is the intersection where the path hit the 1st x plane inside the recon region
		
		if (vertex_x1_y < vertex_x2_y)
        {            
 			alpha_y_1 = (volumn_y * (float)j_min  +boundary_voxel_y - vertex_x1_y )/ (vertex_x2_y - vertex_x1_y);
            j = j_min - 1;
        }
		else if (vertex_x1_y > vertex_x2_y)
        {
 			alpha_y_1 = (volumn_y * (float)j_max  +boundary_voxel_y - vertex_x1_y )/ (vertex_x2_y - vertex_x1_y);
            j = j_max + 1;
        }            
            // Note: alpha_y_1 is the intersection where the path hit the 1st y plane inside the recon region
	
		if (fabs(vertex_x1_z-vertex_x2_z)<volumn_z*1e-6 )  
        {
            alpha_z_1 = MAX_infi;
            k = k_min-1;
        }
        else if (vertex_x1_z < vertex_x2_z)
        {
 			alpha_z_1 = (volumn_z * (float)k_min  +boundary_voxel_z - vertex_x1_z )/ (vertex_x2_z - vertex_x1_z);
            k = k_min - 1;
        }            
		else if (vertex_x1_z > vertex_x2_z)
        {
            alpha_z_1 = (volumn_z * (float)k_max  +boundary_voxel_z - vertex_x1_z )/ (vertex_x2_z - vertex_x1_z);
            k = k_max + 1;
        }        
                                     
        /************ initialization (voxel_i,voxel_j,voxel_k) **************************/            
        // Note: (voxel_i,voxel_j,voxel_k) is the current x,y,z voxel index (@ the initial point at the boundary)
        
        if (vertex_x1_x < vertex_x2_x)
            voxel_i = i_min-1;
        else 
            voxel_i = i_max;
                
        if (vertex_x1_y < vertex_x2_y)
            voxel_j = j_min-1;
        else 
            voxel_j = j_max;
               
        if (fabs(vertex_x1_z-vertex_x2_z) < volumn_z*1e-6)
            voxel_k = k_min-1;
        else if (vertex_x1_z < vertex_x2_z)
            voxel_k = k_min-1;
        else 
            voxel_k = k_max;                              
        
        /***************** Updating alpha_x, alpha_y, alpha_z, ************************/
        
        d_x1_x2 = sqrt((vertex_x2_x-vertex_x1_x)*(vertex_x2_x-vertex_x1_x) + (vertex_x2_y-vertex_x1_y)*(vertex_x2_y - vertex_x1_y) + (vertex_x2_z-vertex_x1_z)*(vertex_x2_z-vertex_x1_z) );
      	        
        alpha_x = alpha_x_1;    // intersection where the path hit the next (i.e. 1st here ) x plane inside the recon
        alpha_y = alpha_y_1;    
        alpha_z = alpha_z_1;    
        
        alpha_c = alpha_min;    // intersection where the path hit the 1st plane at the boundary of recon region

        // Note : (i,j,k) is the (x,y,z) plane index of the current intersection (with a certain plane)
        // If i or j or k should not be an integer, then its predecessor (along the ray)
        
        while (alpha_max - alpha_c > 1e-8)
       	{
            
          if ((voxel_i > M-1)||(voxel_i <0) || (voxel_j > N-1)||(voxel_j <0) || (voxel_k > ZETA-1)||(voxel_k <0))
          {
                alpha_c = alpha_max +1;  // to terminate the loop
          }         
          else
          {
		
  			if ( (alpha_x < alpha_y) && (alpha_x < alpha_z))
                            // alpha_x is the nearest, so update alpha_x
            {
				one_ray_length += d_x1_x2 * (alpha_x - alpha_c);  //(30)
                one_ray_sum += d_x1_x2 * (alpha_x - alpha_c) * d_f[voxel_k*M*N + voxel_j*M + voxel_i];
                                                                //(31)		              
				alpha_c = alpha_x;          //(33)   Update the current location
				N_total_sec = N_total_sec - 1; 
                
                if (vertex_x1_x < vertex_x2_x)
                {
					i = i + 1;
                    voxel_i = voxel_i +1;
                    alpha_x = (volumn_x * (i+1) + boundary_voxel_x - vertex_x1_x )/ (vertex_x2_x - vertex_x1_x);
                }
                if (vertex_x1_x > vertex_x2_x)
                {
                    i = i - 1;      //(29)   
                    voxel_i = voxel_i-1;
                    alpha_x = (volumn_x * (i-1) + boundary_voxel_x - vertex_x1_x )/ (vertex_x2_x - vertex_x1_x);                    
                }
           	}
            
			else if ( (alpha_y < alpha_x) && (alpha_y < alpha_z) )
                            // alpha_y is the nearest, so update alpha_y
           	{                        
				one_ray_length += d_x1_x2 * (alpha_y - alpha_c);
                one_ray_sum += d_x1_x2 * (alpha_y - alpha_c) * d_f[voxel_k*M*N + voxel_j*M + voxel_i];
				
                alpha_c = alpha_y; 
                N_total_sec = N_total_sec -1;                
                
                if (vertex_x1_y < vertex_x2_y)
                {
					j = j + 1;
                    voxel_j = voxel_j+1;
                    alpha_y = (volumn_y * (j+1) + boundary_voxel_y - vertex_x1_y )/ (vertex_x2_y - vertex_x1_y);
                }
                else if (vertex_x1_y > vertex_x2_y)
                {
					j = j - 1;
                    voxel_j = voxel_j-1;
                    alpha_y = (volumn_y * (j-1) + boundary_voxel_y - vertex_x1_y )/ (vertex_x2_y - vertex_x1_y);
                }   

           	}
            
			else if ( (alpha_z < alpha_x) && (alpha_z < alpha_y) )
                        // alpha_z is the nearest, so update alpha_z                
            {				
				one_ray_length += d_x1_x2 * (alpha_z - alpha_c);
                one_ray_sum += d_x1_x2 * (alpha_z - alpha_c) * d_f[voxel_k*M*N + voxel_j*M + voxel_i];
				
                alpha_c = alpha_z; 
                N_total_sec = N_total_sec -1;                
                
                if (vertex_x1_z < vertex_x2_z)
                {
					k = k + 1;
                    voxel_k = voxel_k+1;
                    alpha_z = (volumn_z * (k+1) + boundary_voxel_z - vertex_x1_z )/ (vertex_x2_z - vertex_x1_z);
                }
                else if (vertex_x1_z > vertex_x2_z)
                {
					k = k - 1;
                    voxel_k = voxel_k-1;
                    alpha_z = (volumn_z * (k-1) + boundary_voxel_z - vertex_x1_z )/ (vertex_x2_z - vertex_x1_z);
                }
                
            }
		     
			else if ( (alpha_x == alpha_y) && (alpha_x < alpha_z) )
                        //x = y < z
            {        

				one_ray_length += d_x1_x2 * (alpha_x - alpha_c);  //(30)
                one_ray_sum += d_x1_x2 * (alpha_x - alpha_c) * d_f[voxel_k*M*N + voxel_j*M + voxel_i];  //(31)	
                                                                	              
				alpha_c = alpha_x;          //(33)   Update the current location
				N_total_sec = N_total_sec - 2; 
                                  
                if (vertex_x1_x < vertex_x2_x)
                {
					i = i + 1;
                    voxel_i = voxel_i +1;
                    alpha_x = (volumn_x * (i+1) + boundary_voxel_x - vertex_x1_x )/ (vertex_x2_x - vertex_x1_x);
                }
                if (vertex_x1_x > vertex_x2_x)
                {
                    i = i - 1;      //(29)   
                    voxel_i = voxel_i-1;
                    alpha_x = (volumn_x * (i-1) + boundary_voxel_x - vertex_x1_x )/ (vertex_x2_x - vertex_x1_x);                    
                }
                
                if (vertex_x1_y < vertex_x2_y)
                {
					j = j + 1;
                    voxel_j = voxel_j+1;
                    alpha_y = (volumn_y * (j+1) + boundary_voxel_y - vertex_x1_y )/ (vertex_x2_y - vertex_x1_y);
                }
                else if (vertex_x1_y > vertex_x2_y)
                {
					j = j - 1;
                    voxel_j = voxel_j-1;
                    alpha_y = (volumn_y * (j-1) + boundary_voxel_y - vertex_x1_y )/ (vertex_x2_y - vertex_x1_y);
                }                
                
            }
            
          	else if ( (alpha_x == alpha_z) && (alpha_x < alpha_y))// && (sphere_range<=1.0f) )
                        // x = z < y;
            {                      
				one_ray_length += d_x1_x2 * (alpha_x - alpha_c);  //(30)
                one_ray_sum += d_x1_x2 * (alpha_x - alpha_c) * d_f[voxel_k*M*N + voxel_j*M + voxel_i];  //(31)	
                                                                	              
				alpha_c = alpha_x;          //(33)   Update the current location
				N_total_sec = N_total_sec - 2; 

                if (vertex_x1_x < vertex_x2_x)
                {
					i = i + 1;
                    voxel_i = voxel_i +1;
                    alpha_x = (volumn_x * (i+1) + boundary_voxel_x - vertex_x1_x )/ (vertex_x2_x - vertex_x1_x);
                }
                if (vertex_x1_x > vertex_x2_x)
                {
                    i = i - 1;      //(29)   
                    voxel_i = voxel_i-1;
                    alpha_x = (volumn_x * (i-1) + boundary_voxel_x - vertex_x1_x )/ (vertex_x2_x - vertex_x1_x);                    
                }
                
                if (vertex_x1_z < vertex_x2_z)
                {
					k = k + 1;
                    voxel_k = voxel_k+1;
                    alpha_z = (volumn_z * (k+1) + boundary_voxel_z - vertex_x1_z )/ (vertex_x2_z - vertex_x1_z);
                }
                else if (vertex_x1_z > vertex_x2_z)
                {
					k = k - 1;
                    voxel_k = voxel_k-1;
                    alpha_z = (volumn_z * (k-1) + boundary_voxel_z - vertex_x1_z )/ (vertex_x2_z - vertex_x1_z);
                }                

            }
            
			else if ( (alpha_y == alpha_z) && (alpha_y < alpha_x))// && (sphere_range<=1.0f) )
                      	// y = z < x        	
            {            	
				one_ray_length += d_x1_x2 * (alpha_y - alpha_c);
                one_ray_sum += d_x1_x2 * (alpha_y - alpha_c) * d_f[voxel_k*M*N + voxel_j*M + voxel_i];
				
                alpha_c = alpha_y; 
                N_total_sec = N_total_sec -2;                
                
                if (vertex_x1_y < vertex_x2_y)
                {
					j = j + 1;
                    voxel_j = voxel_j+1;
                    alpha_y = (volumn_y * (j+1) + boundary_voxel_y - vertex_x1_y )/ (vertex_x2_y - vertex_x1_y);
                }
                else if (vertex_x1_y > vertex_x2_y)
                {
					j = j - 1;
                    voxel_j = voxel_j-1;
                    alpha_y = (volumn_y * (j-1) + boundary_voxel_y - vertex_x1_y )/ (vertex_x2_y - vertex_x1_y);
                }   
                
                if (vertex_x1_z < vertex_x2_z)
                {
					k = k + 1;
                    voxel_k = voxel_k+1;
                    alpha_z = (volumn_z * (k+1) + boundary_voxel_z - vertex_x1_z )/ (vertex_x2_z - vertex_x1_z);
                }
                else if (vertex_x1_z > vertex_x2_z)
                {
					k = k - 1;
                    voxel_k = voxel_k-1;
                    alpha_z = (volumn_z * (k-1) + boundary_voxel_z - vertex_x1_z )/ (vertex_x2_z - vertex_x1_z);
                }         
                
            }
			
          	else if ( (alpha_x == alpha_z) && (alpha_x == alpha_y))// && (sphere_range<=1.0f) )
                        // x=y=z            
            {
				one_ray_length += d_x1_x2 * (alpha_x - alpha_c);  //(30)
                one_ray_sum += d_x1_x2 * (alpha_x - alpha_c) * d_f[voxel_k*M*N + voxel_j*M + voxel_i];  //(31)	
                                                                	              
				alpha_c = alpha_x;          //(33)   Update the current location
				N_total_sec = N_total_sec - 3; 


                if (vertex_x1_x < vertex_x2_x)
                {
					i = i + 1;
                    voxel_i = voxel_i +1;
                    alpha_x = (volumn_x * (i+1) + boundary_voxel_x - vertex_x1_x )/ (vertex_x2_x - vertex_x1_x);
                }
                if (vertex_x1_x > vertex_x2_x)
                {
                    i = i - 1;      //(29)   
                    voxel_i = voxel_i-1;
                    alpha_x = (volumn_x * (i-1) + boundary_voxel_x - vertex_x1_x )/ (vertex_x2_x - vertex_x1_x);                    
                }
                
                if (vertex_x1_y < vertex_x2_y)
                {
					j = j + 1;
                    voxel_j = voxel_j+1;
                    alpha_y = (volumn_y * (j+1) + boundary_voxel_y - vertex_x1_y )/ (vertex_x2_y - vertex_x1_y);
                }
                else if (vertex_x1_y > vertex_x2_y)
                {
					j = j - 1;
                    voxel_j = voxel_j-1;
                    alpha_y = (volumn_y * (j-1) + boundary_voxel_y - vertex_x1_y )/ (vertex_x2_y - vertex_x1_y);
                }   
                
                if (vertex_x1_z < vertex_x2_z)
                {
					k = k + 1;
                    voxel_k = voxel_k+1;
                    alpha_z = (volumn_z * (k+1) + boundary_voxel_z - vertex_x1_z )/ (vertex_x2_z - vertex_x1_z);
                }
                else if (vertex_x1_z > vertex_x2_z)
                {
					k = k - 1;
                    voxel_k = voxel_k-1;
                    alpha_z = (volumn_z * (k-1) + boundary_voxel_z - vertex_x1_z )/ (vertex_x2_z - vertex_x1_z);
                }  
           	}
          }
       	}// end while
        
        
        if (one_ray_length < volumn_z*1e-6)            
            d_proj_correction[proj_pixel_index] = 0.0;
        else
        {
            if (command==0) 
              	d_proj_correction[proj_pixel_index] = one_ray_sum; // forward operator
            else if (command==1) 
                d_proj_correction[proj_pixel_index] = (d_proj_data[proj_pixel_index] - one_ray_sum)/one_ray_length; 
                                                                    // projection correction (for SART)
        }
        if (Error > 0.0f)
            d_proj_correction[proj_pixel_index] = Error;   // Check Error Message        
        
     }//else if 
   }//else if

//    __syncthreads();
}



__global__ void backprj_ray_driven_3d_kernel(float *d_volumn_kernel, float *d_proj_correction, float beta_temp, float sin_theta, float cos_theta, int command)
{    
    /* 
     * Reference: "Accelerating simultaneous algebraic reconstruction technique with motion compensation using CUDA-enabled GPU" 
     * Wai-Man Pang, CUHK
     * Section: Back-projection and image update
     
     * d_proj_correction : 2D projection correction, i.e. c(i) in the Wai-Man Pang, CUHK paper
     * t_theta : projection angle
     * beta_temp : lamda in the paper
     * d_volumn: 3D object array
     * d_volumn(j) = d_volumn(j) + beta_temp * sum_i (c(i)*w(ij)) / sum_i (w(ij));  where i is ray index, j is voxel index
     */        
    
    /* Note
     * dim3  dimGrid_backprj(N,ZETA,N_source);  =>
     *  blockIdx.x ranges 0~(N-1), index of pixels along y axis
     *  blockIdx.y ranges 0~(ZETA-1), index of pixels along z axis
     *  blockIdx.z ranges 0~(N_source-1), index of source
     * dim3  dimBlock_backprj(M);  =>
     *  threadIdx.x ranges 0~(M-1), index of pixels along x axis
     * blockDim.x = M, gridDim.x=N, gridDim.y=ZETA
     */
    
//     int Idx_image_x = threadIdx.x;
    int Idx_image_x = threadIdx.x + blockIdx.x/256*blockDim.x;    
//     int Idx_image_y = blockIdx.x;
    int Idx_image_y = blockIdx.x % 256 ;    
    int Idx_image_z = blockIdx.y;
    int image_voxel_index = Idx_image_z*M*N + Idx_image_y*M + Idx_image_x;  
    
	//coordinate of center of each voxel in x-y-z system 
	float coord_voxel_x = boundary_voxel_x + volumn_x*(0.5f + Idx_image_x);
    float coord_voxel_y = boundary_voxel_y + volumn_y*(0.5f + Idx_image_y);
	float coord_voxel_z = boundary_voxel_z + volumn_z*(0.5f + Idx_image_z);  
    
    /**************************************/    
	float coord_vertex_s, coord_vertex_t;

	float minY = MAX_infi, minZ=MAX_infi, maxY=-MAX_infi, maxZ=-MAX_infi;

	float coord_source_x, coord_source_y, coord_source_z;
	float alpha_x_i_1, alpha_x_i;
	float alpha_y_i_1, alpha_y_i;
	float alpha_z_i_1, alpha_z_i;
   	float alpha_x_temp, alpha_y_temp, alpha_z_temp; 
	float alpha_min, alpha_max; 
	
	int minY_index, maxY_index, minZ_index, maxZ_index; 
	float sumLength=0.0f, sumWeight=0.0f;
	float d_x1_x2; 
	
//     float weight = 1.0f;
//     float tao;
//     float tao_m1 = atan( (float(R)*Detector_pixel_x/2.0f-abs(Offset)) / DSO); 
    
    float Error = 0.0f;
    
    /***********************************************************/
    
	if ( (Idx_image_x-(float(M)/2.0f-0.5)-M_Offset)*volumn_x*(Idx_image_x-(float(M)/2.0f-0.5)-M_Offset)*volumn_x 
            +  (Idx_image_y - (float(N)/2.0f-0.5))*volumn_y*(Idx_image_y-(float(N)/2.0f-0.5))*volumn_y 
            >= (float(M)/2.0f-0.5)*volumn_y*(float(N)/2.0f-0.5)*volumn_y )
        
         d_volumn_kernel[image_voxel_index]  = 0.0f ;

	else
            // Note: The following codes apply to all the voxels simutaneously
	{
        
        // source coordinate in (x,y,z) system after rotation
        coord_source_x = Source_x * cos_theta - Source_y * sin_theta;
        coord_source_y = Source_x * sin_theta + Source_y * cos_theta;
        coord_source_z = Source_z_min + blockIdx.z * Source_interval;
        
        
        /******** exclude some sources ********/
        
		coord_vertex_t = coord_voxel_x * cos_theta + coord_voxel_y * sin_theta; 
		coord_vertex_s = - coord_voxel_x * sin_theta + coord_voxel_y * cos_theta;	   
        
        float source_upper = (coord_voxel_z+Detector_pixel_x-Detector_Zmin)*(DSO-DOD)/(coord_vertex_t-DOD)+Detector_Zmin;
        float source_lower = (coord_voxel_z-Detector_pixel_x-Detector_Zmax)*(DSO-DOD)/(coord_vertex_t-DOD)+Detector_Zmax;
        
        if (coord_source_z+Source_interval*1<source_lower || coord_source_z-Source_interval*1>source_upper)
            return;

        /******** investigate the eight vertices of each voxel ********/
        
        for (int k=0;k<2;k++)
            for (int j=0;j<2;j++)
                for (int i=0;i<2;i++)
		{

			//coordinate for each of eight vertices of the voxel 
			float coord_vertex_x = coord_voxel_x + float(i)*volumn_x - 0.5f*volumn_x; 
			float coord_vertex_y = coord_voxel_y + float(j)*volumn_y - 0.5f*volumn_y; 
			float coord_vertex_z = coord_voxel_z + float(k)*volumn_z - 0.5f*volumn_z; 
            
			// <x-y> <----> <t-s>
			coord_vertex_t = coord_vertex_x * cos_theta + coord_vertex_y * sin_theta; 
			coord_vertex_s = - coord_vertex_x * sin_theta + coord_vertex_y * cos_theta;			
            // Note: transformation from Cartesian Coordiantor to rotated coordiator with angle of t_theta
            // In the new coordiantor, detector plane is prependicular to the t axis 
			
            // in <t,s> system            
			float coord_vertexOnDetector_x = (coord_vertex_t - DOD)/(DSO- coord_vertex_t) * (coord_vertex_s - Source_y)+ coord_vertex_s ;
			float coord_vertexOnDetector_z = (coord_vertex_t - DOD)/(DSO- coord_vertex_t) * (coord_vertex_z - coord_source_z)+ coord_vertex_z ;
            // the projcetion of the vertex of the voxel

			minY= fmin(minY, coord_vertexOnDetector_x);
			maxY= fmax(maxY, coord_vertexOnDetector_x); 
			minZ= fmin(minZ, coord_vertexOnDetector_z);
			maxZ= fmax(maxZ, coord_vertexOnDetector_z); 
            // form a minimim bounding rectangle (MBR) for these vertexes
            
		}

        minY_index = floor( (minY -  Detector_Ymin ) / Detector_pixel_x +0.5f);
        maxY_index = floor( (maxY -  Detector_Ymin ) / Detector_pixel_x +0.5f);
        minZ_index = floor( (minZ -  Detector_Zmin ) / Detector_pixel_x +0.5f);
        maxZ_index = floor( (maxZ -  Detector_Zmin ) / Detector_pixel_x +0.5f);
        // index of pixels of MBR boudaries on the detector 
        
        /***********************************/

        // If this voxel does not project on this detector plane, it means there is no ray passing throught this voxel at this angle.
        if ( (minY_index<0) && (maxY_index <0) )
        {	
            d_volumn_kernel[image_voxel_index]  += 0.0f ;  
        }
        else if ( (minY_index>(R-1)) && (maxY_index >(R-1)) )
        {	
            d_volumn_kernel[image_voxel_index]  += 0.0f ; 
        }
        else if ( (minZ_index<0) && (maxZ_index <0 ) )
        {	
            d_volumn_kernel[image_voxel_index]  += 0.0f ;  		
        }
        else if ( (minZ_index>(Z_prj-1)) && (maxZ_index >(Z_prj -1)) )
        {	
            d_volumn_kernel[image_voxel_index]  += 0.0f ;  		
        }
        
        else            
            // If this voxel projects on the detector plane 
        {
            
	    	if (minY_index <=0)
        	        minY_index = 0;
        	if (maxY_index >=(R-1) )
                	maxY_index = R-1;
        	if (minZ_index <=0)
                	minZ_index = 0;
        	if (maxZ_index >=(Z_prj-1) )
                	maxZ_index = Z_prj-1;


            // for those projection pixels whose coordinate loacates inside MBR
            // Each pixel coorresponds to a ray, and that ray must pass through the specific voxel
            for (int j=minZ_index; j<=maxZ_index; j++) 
                for (int i=minY_index; i<=maxY_index; i++)
            {
                float coord_pixelOnDetector_x =  DOD * cos_theta - (Detector_Ymin + i*Detector_pixel_x) * sin_theta ;
                float coord_pixelOnDetector_y =  DOD * sin_theta + (Detector_Ymin + i*Detector_pixel_x) * cos_theta ;
                float coord_pixelOnDetector_z =  Detector_Zmin + j*Detector_pixel_x;
                // coordinate of the the pixel inside MBR in (x,y,z) system                
                
                /** Weighted Update for Half Detector **/
//                 if ( (float(i)*Detector_pixel_x) < 2.0f*abs(Offset) )
//                     weight = 1.0f;                
//                 else
//                 {
//                     tao = atan( ( float(R/2-i)*Detector_pixel_x + abs(Offset) ) / DSO);                    
//                     weight = cos(PI/4*(tao/tao_m1 - 1));
//                     weight = weight * weight;                     
//                 }
                /******/  
                
                
                // Next: investigate the line starting at x1 and ending at x2 
                // find out all the rays whose projection lies in the rectangle.

                if ( (coord_source_x != coord_pixelOnDetector_x) && (coord_source_y != coord_pixelOnDetector_y) && (coord_source_z != coord_pixelOnDetector_z) )
                    // in fact, I carefully design that coord_source_x != coord_pixelOnDetector_x; 
                {	
                    alpha_x_i_1 =  ( (coord_voxel_x - 0.5f*volumn_x) - coord_pixelOnDetector_x )/( coord_source_x - coord_pixelOnDetector_x ); 
                    alpha_x_i   =  ( (coord_voxel_x + 0.5f*volumn_x) - coord_pixelOnDetector_x )/( coord_source_x - coord_pixelOnDetector_x ); 
                    alpha_y_i_1 =  ( (coord_voxel_y - 0.5f*volumn_y) - coord_pixelOnDetector_y )/( coord_source_y - coord_pixelOnDetector_y ); 
                    alpha_y_i   =  ( (coord_voxel_y + 0.5f*volumn_y) - coord_pixelOnDetector_y )/( coord_source_y - coord_pixelOnDetector_y );
                    alpha_z_i_1 =  ( (coord_voxel_z - 0.5f*volumn_z) - coord_pixelOnDetector_z )/( coord_source_z - coord_pixelOnDetector_z ); 
                    alpha_z_i   =  ( (coord_voxel_z + 0.5f*volumn_z) - coord_pixelOnDetector_z )/( coord_source_z - coord_pixelOnDetector_z ); 
                    // find out indices of the two most closet x planes near this specific voxel

                    alpha_x_temp = fmin((alpha_x_i_1), (alpha_x_i));
                    alpha_y_temp = fmin((alpha_y_i_1), (alpha_y_i)); 
                    if (fabs(coord_source_z - coord_pixelOnDetector_z) < volumn_z*1e-3) 
                        alpha_z_temp = -MAX_infi;                    
                    else
                        alpha_z_temp = fmin((alpha_z_i_1), (alpha_z_i)); 
					alpha_min = fmax(fmax(alpha_x_temp, alpha_y_temp), fmax(alpha_y_temp, alpha_z_temp)); 
                    // get the maximum value of the three minimum values. so the alpha_min is the enter point for one specific voxel
		
                    alpha_x_temp = fmax((alpha_x_i_1), (alpha_x_i));
                    alpha_y_temp = fmax((alpha_y_i_1), (alpha_y_i));
                    if (fabs(coord_source_z - coord_pixelOnDetector_z) < volumn_z*1e-3) 
                        alpha_z_temp = MAX_infi;                    
                    else                    
                        alpha_z_temp = fmax((alpha_z_i_1), (alpha_z_i));
					alpha_max = fmin(fmin(alpha_x_temp, alpha_y_temp), fmin(alpha_y_temp, alpha_z_temp));
                    // alpha_max is the exit point of the line passing through this voxel
			
                    d_x1_x2 = sqrt((coord_source_x-coord_pixelOnDetector_x)*(coord_source_x-coord_pixelOnDetector_x) + (coord_source_y-coord_pixelOnDetector_y)*(coord_source_y - coord_pixelOnDetector_y) + (coord_source_z-coord_pixelOnDetector_z)*(coord_source_z-coord_pixelOnDetector_z) );
                    
                    if  ( d_x1_x2*(alpha_max-alpha_min) > volumn_z*1e-6)
                        // the line passes through the voxel with a sufficient length; 
                        // if the value is negative, it means the ray does not pass through this voxel
                    {
                        sumWeight  = sumWeight +  d_x1_x2 * (alpha_max - alpha_min)*d_proj_correction[blockIdx.z*R*Z_prj + j*R  + i];
                        // Note: d_proj_correction[j*R + i] is c(i) which has been previously calculated
                        // Note: d_x1_x2 * (alpha_max - alpha_min) is w(i) for ray i of this projection 
                        sumLength = sumLength +  (alpha_max - alpha_min)*d_x1_x2;	
                        
                    }
			
                }
            }// end for loop: all the rays whose projection fits in the rectangle

            if (sumLength < volumn_x*1e-3)
            	d_volumn_kernel[image_voxel_index] += 0.0f ;  
            else
            {
                if (command==0)
                    d_volumn_kernel[image_voxel_index] += beta_temp * sumWeight/sumLength ;  
                else if (command==1)
                    d_volumn_kernel[image_voxel_index] += beta_temp * sumWeight ;   // matched ajoint operator, for test use             
            }
            
            if (Error > 0.0f)
                d_volumn_kernel[image_voxel_index] = Error ;            // Check Error Message


                    
        }//end else if this voxel projects on this detector plane 
        
    }//end else if the reconstruction region is in the circle
    
//     __syncthreads();    
}
 

__global__ void reduce_norm_2_kernel_l1(float *g_idata, float *g_odata, unsigned int n)
{
 	
	//load shared_mem
	extern __shared__ float sdata[];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.y* blockDim.x*gridDim.x + blockIdx.x*blockDim.x + threadIdx.x;

	sdata[tid] = (i < n) ? (g_idata[i]*g_idata[i]) : 0;

	__syncthreads();
	// do reduction in shared mem
	for(unsigned int s=blockDim.x/2; s>0; s>>=1)
        {
            if (tid < s)
                {
                        sdata[tid] += sdata[tid + s];
                }
            __syncthreads();
        }
    	// write result for this block to global mem
	if (tid == 0) g_odata[blockIdx.y*gridDim.x + blockIdx.x]  = sdata[0];
}

__global__ void reduce_norm_tv_kernel_l1(float *g_idata, float *g_odata, unsigned int n)
{
 	
	//load shared_mem
	extern __shared__ float sdata[];
        unsigned int tid = threadIdx.x;
        unsigned int i = blockIdx.y* blockDim.x*gridDim.x + blockIdx.x*blockDim.x + threadIdx.x;

	sdata[tid] = (i < n) ? (g_idata[i]) : 0;

        __syncthreads();
        // do reduction in shared mem
	for(unsigned int s=blockDim.x/2; s>0; s>>=1)
        {
                if (tid < s)
                {
                        sdata[tid] += sdata[tid + s];
                }
        __syncthreads();
        }
    	// write result for this block to global mem
	if (tid == 0) g_odata[blockIdx.y*gridDim.x + blockIdx.x]  = sdata[0];
}

__global__ void reduce_norm_2_kernel_l2(float *g_idata, float *g_odata, unsigned int n)
{

	//load shared mem 
	extern __shared__ float sdata[];
        unsigned int tid = threadIdx.x;
        unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

         sdata[tid] = (i < n) ? fabs(g_idata[i]) : 0;

        __syncthreads();
	// do reduction in shared mem
        for(unsigned int s=blockDim.x/2; s>0; s>>=1)
        {
                if (tid < s)
                {
                        sdata[tid] += sdata[tid + s];
                }
        __syncthreads();
        }
	// write result for this block to global mem
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];

}

__global__ void reduce_norm_2_kernel_end(float *g_idata, float *g_odata, unsigned int n)
{
 
        extern __shared__ float sdata[];
        unsigned int tid = threadIdx.x;

         sdata[tid] = (tid < n) ? fabs(g_idata[tid]) : 0;

        __syncthreads();
        // do reduction in shared mem
        for(unsigned int s=blockDim.x/2; s>0; s>>=1)
        {
                if (tid < s)
                {
                        sdata[tid] += sdata[tid + s];
                }
        __syncthreads();
        }
	
	// write result for this block to global mem
        if (tid == 0) g_odata[0] = sqrt(sdata[0]);

}


__global__ void tv_gradient_matrix_3d_kernel(float *df, float *d_volumn, float epi)
{
        
	int t_id, bx_id, by_id;
        t_id  = threadIdx.x+1;
        bx_id = blockIdx.x+1;
        by_id = blockIdx.y+1;
        float stl, s_sub_1_tl, s_t_sub_1_l, st_l_sub_1;
        float s_add_1_tl, s_add_1_t_sub_1_l, s_add_1_t_l_sub_1;
        float s_t_add_1_l, s_sub_1_t_add_1_l, s_t_add_1_l_sub_1;
        float st_l_add_1, s_sub_1_t_l_add_1, s_t_sub_1_l_add_1;

        stl             = d_volumn[by_id*N*M + bx_id*M + t_id];
        s_sub_1_tl      = d_volumn[(by_id-1)*N*M + bx_id*M + t_id];
        s_t_sub_1_l     = d_volumn[by_id*N*M + (bx_id-1)*M + t_id];
        st_l_sub_1      = d_volumn[by_id*N*M + bx_id*M + t_id-1];

        s_add_1_tl      = d_volumn[(by_id+1)*N*M + bx_id*M + t_id];
        s_add_1_t_sub_1_l =  d_volumn[(by_id+1)*N*M + (bx_id-1)*M + t_id];
        s_add_1_t_l_sub_1 =  d_volumn[(by_id+1)*N*M + bx_id*M + t_id-1];

        s_t_add_1_l     = d_volumn[by_id*N*M + (bx_id+1)*M + t_id];
        s_sub_1_t_add_1_l = d_volumn[(by_id-1)*N*M + (bx_id+1)*M + t_id];
        s_t_add_1_l_sub_1 = d_volumn[by_id*N*M + (bx_id+1)*M + t_id-1];

        st_l_add_1      =d_volumn[by_id*N*M + bx_id*M + t_id + 1];
        s_sub_1_t_l_add_1 = d_volumn[(by_id-1)*N*M + bx_id*M + t_id + 1];
        s_t_sub_1_l_add_1 = d_volumn[by_id*N*M + (bx_id-1)*M + t_id + 1];

        df[by_id*N*M + bx_id*M + t_id] = ((stl - s_sub_1_tl) + (stl - s_t_sub_1_l) + (stl - st_l_sub_1) ) /sqrt(epi +  (stl - s_sub_1_tl)* (stl - s_sub_1_tl) + (stl - s_t_sub_1_l)* (stl - s_t_sub_1_l) +   (stl - st_l_sub_1)* (stl - st_l_sub_1) )
        - (s_add_1_tl - stl)/sqrt(epi +  (s_add_1_tl - stl)*(s_add_1_tl - stl)  +  (s_add_1_tl - s_add_1_t_sub_1_l)*(s_add_1_tl - s_add_1_t_sub_1_l) + (s_add_1_tl - s_add_1_t_l_sub_1)*(s_add_1_tl - s_add_1_t_l_sub_1))

        - (s_t_add_1_l - stl)/sqrt(epi +  (s_t_add_1_l - s_sub_1_t_add_1_l)*(s_t_add_1_l - s_sub_1_t_add_1_l) + (s_t_add_1_l - stl)*(s_t_add_1_l - stl) + (s_t_add_1_l - s_t_add_1_l_sub_1)* (s_t_add_1_l - s_t_add_1_l_sub_1))

        - (st_l_add_1 - stl)/sqrt(epi +  (st_l_add_1 - s_sub_1_t_l_add_1)*(st_l_add_1 - s_sub_1_t_l_add_1) + (st_l_add_1 - s_t_sub_1_l_add_1)*(st_l_add_1 - s_t_sub_1_l_add_1) + (st_l_add_1 - stl)* (st_l_add_1 - stl));


}


__global__ void tv_matrix_3d_kernel(float *df, float *d_volumn)
{
        
	int t_id, bx_id, by_id;
        t_id  = threadIdx.x+1;
        bx_id = blockIdx.x+1;
        by_id = blockIdx.y+1;
       
	float stl, s_sub_1_tl, s_t_sub_1_l, st_l_sub_1;

        stl             = d_volumn[by_id*N*M + bx_id*M + t_id];
        s_sub_1_tl      = d_volumn[(by_id-1)*N*M + bx_id*M + t_id];
        s_t_sub_1_l     = d_volumn[by_id*N*M + (bx_id-1)*M + t_id];
        st_l_sub_1      = d_volumn[by_id*N*M + bx_id*M + t_id-1];

	df[by_id*N*M + bx_id*M + t_id] = sqrt( (stl - s_sub_1_tl)*(stl - s_sub_1_tl) + (stl - s_t_sub_1_l)*(stl - s_t_sub_1_l) + (stl - st_l_sub_1)*(stl - st_l_sub_1)) ;

}

__global__ void backtracking_update_kernel(float *d_volumn_f_update,float *d_volumn_f, float *d_tv_gradient_matrix ,float alpha_temp)
{
     
        unsigned int i = blockIdx.y* blockDim.x*gridDim.x + blockIdx.x*blockDim.x + threadIdx.x;	
	d_volumn_f_update[i] = d_volumn_f[i] -  alpha_temp*d_tv_gradient_matrix[i];
}


