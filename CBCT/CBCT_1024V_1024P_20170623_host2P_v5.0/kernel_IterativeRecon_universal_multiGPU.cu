#include "hip/hip_runtime.h"
__global__ void kernel_add_proj(float *d_a, float *d_b)
{
	int idx = blockDim.x * gridDim.x * blockIdx.y +  blockDim.x * blockIdx.x + threadIdx.x; 
    d_a[idx]=d_a[idx]+d_b[idx];
}

__global__ void kernel_divide_proj(float *h_proj_correction, float *h_proj_data, float *h_proj_sumLen, float *h_proj_weightedLen)
{
	int idx = blockDim.x * gridDim.x * blockIdx.y +  blockDim.x * blockIdx.x + threadIdx.x; 
    
    float temp = h_proj_sumLen[idx];
    
    if ( temp < volumn_z*1e-6)
        h_proj_correction[idx] = 0;
    else
    {        
        h_proj_correction[idx] = (h_proj_data[idx] - h_proj_weightedLen[idx]) / temp ;
    }
}

__global__ void forward_ray_driven_3d_kernel_correction_multiGPU(float *d_f , float *d_proj_correction, float *d_proj_data, float sin_theta, float cos_theta, int subPrjIdx, int command)

{
	// d_f: 3D object array;    d_f[i,j,k] = d_f [k*M*N+j*M+i]; 
    // d_proj_data: 2D projection acquired at the angle of t_theta (only a portion of the whole projection view)
	// d_proj_correction: 2D projection correction,  (output of this function. i.e. c(i) in the paper)    
    // subPrjIdx: sub projection portion index
                
    int Detector_x_idx = threadIdx.x + blockDim.x * blockIdx.x;
    int Detector_z_idx = blockIdx.y;
    
    int proj_pixel_index = Detector_z_idx * R + Detector_x_idx;
    
	// Source position (X2): coordinate in (x,y,z) system . 
    float vertex_x2_x,vertex_x2_y,vertex_x2_z;    
    if (CT_style==0)   //CBCT
    {
        vertex_x2_x = Source_x * cos_theta - Source_y * sin_theta;
        vertex_x2_y = Source_x * sin_theta + Source_y * cos_theta;
        vertex_x2_z = Source_z;
    }
    else if (CT_style==1) //FBCT
    {
        vertex_x2_x = Source_x * cos_theta - Source_y * sin_theta;
        vertex_x2_y = Source_x * sin_theta + Source_y * cos_theta;
        vertex_x2_z = Detector_Zmin + (Z_prj/Number_of_Devices*subPrjIdx+Detector_z_idx) * Detector_pixel_x; 
    }    
    else if (CT_style==2) //parallel beam
    {
        vertex_x2_x = Source_x * cos_theta - (Detector_Ymin +  Detector_x_idx * Detector_pixel_x) * sin_theta;
        vertex_x2_y = Source_x * sin_theta + (Detector_Ymin +  Detector_x_idx * Detector_pixel_x) * cos_theta;
        vertex_x2_z = Detector_Zmin + (Z_prj/Number_of_Devices*subPrjIdx+Detector_z_idx) * Detector_pixel_x;        
    }
    
    
    // Detector element center positions (X1): Coordinate in (x,y,z) system --- 
    float vertex_x1_x,vertex_x1_y,vertex_x1_z;
    vertex_x1_x = DOD * cos_theta - (Detector_Ymin +  Detector_x_idx * Detector_pixel_x) * sin_theta;
    vertex_x1_y = DOD * sin_theta + (Detector_Ymin +  Detector_x_idx * Detector_pixel_x) * cos_theta;
    vertex_x1_z = Detector_Zmin + (Z_prj/Number_of_Devices*subPrjIdx+Detector_z_idx) * Detector_pixel_x;        
        
    //  Notice: in this system, vertex_x1_x < 0 < vertex_x2_x    
            
    float inv_x_diff = 1.0f / (vertex_x2_x - vertex_x1_x);
    float inv_y_diff = 1.0f / (vertex_x2_y - vertex_x1_y);
    float inv_z_diff = 1.0f / (vertex_x2_z - vertex_x1_z);    
    
    /*****************************************/
                
    float alpha_x_min= 0.0f, alpha_y_min= 0.0f, alpha_z_min= 0.0f;
    float alpha_x_max= 0.0f, alpha_y_max= 0.0f, alpha_z_max= 0.0f;
    float alpha_min= 0.0f, alpha_max= 0.0f;
        
	int i_min=0, j_min=0, k_min=0;
	int i_max=0, j_max=0, k_max=0;
    int i=0, j=0, k=0;
    int voxel_i=0, voxel_j=0, voxel_k=0;
    
	float alpha_x=0.0f, alpha_y=0.0f, alpha_z=0.0f;  
    float one_ray_sum = 0.0f;
    float one_ray_length = 0.0f; 

    float alpha_c= 0.0f;
    float d_x1_x2= 0.0f;

	int N_total_sec=0; 
    
    int next_alpha_index;

            
	/**** Step 1 :find out alpha_min, alpha_max ********/

    
	alpha_min = (boundary_voxel_x + volumn_x*0 - vertex_x1_x )* inv_x_diff; //(9)
    alpha_max = (boundary_voxel_x + volumn_x*M - vertex_x1_x )* inv_x_diff;
        // Notice: it is still unsure here which one is the parametric value of the first intersection point of the ray with the x-plane
        // It depends on whether source or detector lies on the left side of the reconstruction region at this time

    alpha_x_min = fmin(alpha_min, alpha_max);   //(5)
    alpha_x_max = fmax(alpha_min, alpha_max );  //(6) 
                
    alpha_min = (boundary_voxel_y + volumn_y*0 - vertex_x1_y )* inv_y_diff;
    alpha_max = (boundary_voxel_y + volumn_y*N - vertex_x1_y )* inv_y_diff;

    alpha_y_min = fmin(alpha_min, alpha_max);   //(7)
    alpha_y_max = fmax(alpha_min, alpha_max );  //(8)
        
    alpha_min = (boundary_voxel_z + volumn_z*0 - vertex_x1_z )* inv_z_diff;
    alpha_max = (boundary_voxel_z + volumn_z*ZETA - vertex_x1_z )* inv_z_diff;
    // Note: when (vertex_x2_z == vertex_x1_z), alpha_min = -inf, alpha_max = inf.
        
    alpha_z_min = fmin(alpha_min, alpha_max);   
    alpha_z_max = fmax(alpha_min, alpha_max );  
    
        // alpha_min / alpha_max reused 
    alpha_min = fmax(fmax(alpha_x_min, alpha_y_min), fmax(alpha_y_min, alpha_z_min)); //(3)
        // i.e. alpha_min = fmax(alpha_x_min,alpha_y_min,alpha_z_min)
        // it indicates the point where the path interacts with the near boundary of reconstruction region        

    alpha_max = fmin(fmin(alpha_x_max, alpha_y_max), fmin(alpha_y_max, alpha_z_max)); //(4)
        // i.e. alpha_max = fmin(alpha_x_max,alpha_y_max,alpha_z_max)
        // it indicates the point where the path last interacts with the far boundary of reconstruction region        
        
        /********Step 2,3: Find i_max, i_min***************/
        
     if (alpha_max <= alpha_min)   // It means no interaction of the ray and the volume
            one_ray_length = 0.0f ;
  
	 else 
     {
			// X direction 
			if (vertex_x1_x < vertex_x2_x)
			{	
				if (alpha_min == alpha_x_min)
					i_min = 1;      //(11)
				else //if (alpha_min != alpha_x_min)
					i_min =  floor(( alpha_min*(vertex_x2_x - vertex_x1_x) + vertex_x1_x - boundary_voxel_x)*inv_volumn_x) + 1 ;
                                    //(12)
                     /* Note: i_min is the index of the 1st x plane where the path interacts inside the reconstruction region
                      * It is not the index of alpha_x_min
                      */                
				if (alpha_max == alpha_x_max)
					i_max = M;      //(13)
				else //if (alpha_max != alpha_x_max)
					i_max =  floor(( alpha_max*(vertex_x2_x - vertex_x1_x) + vertex_x1_x - boundary_voxel_x)*inv_volumn_x) ;
                                    //(14)
                     // Note: i_max is the index of the last x plane where the path interacts with the reconstruction region (inside or boundary)                      
			}	
			else //if (vertex_x1_x >= vertex_x2_x)
			{	
				if (alpha_min == alpha_x_min)
					i_max = M-1;    //(15)
				else //if (alpha_min != alpha_x_min)
					i_max =  floor(( alpha_min*(vertex_x2_x - vertex_x1_x) + vertex_x1_x - boundary_voxel_x)*inv_volumn_x) ;				
                                    //(16)
				if (alpha_max == alpha_x_max)
					i_min = 0;      //(17)
				else //if (alpha_max != alpha_x_max)
					i_min =  floor(( alpha_max*(vertex_x2_x - vertex_x1_x) + vertex_x1_x - boundary_voxel_x)*inv_volumn_x) + 1 ;
                                    //(18)
			}	
            // Note: overall, i_min is the most left x-plane, i_max the most right x-plane,
            // and the initial point (the first interacted position on the boundary) NOT included.            
               
			//Y direction 
			if (vertex_x1_y < vertex_x2_y)
			{	
				if (alpha_min == alpha_y_min)
					j_min = 1; 
				else //f (alpha_min != alpha_y_min)
					j_min =  floor(( alpha_min*(vertex_x2_y - vertex_x1_y) + vertex_x1_y - boundary_voxel_y)*inv_volumn_y) + 1 ;
				
				if (alpha_max == alpha_y_max)
					j_max = N; 
				else //if (alpha_max != alpha_y_max)
					j_max =  floor(( alpha_max*(vertex_x2_y - vertex_x1_y) + vertex_x1_y - boundary_voxel_y)*inv_volumn_y) ;

			}	
			else //if (vertex_x1_y >= vertex_x2_y)
			{	
				if (alpha_min == alpha_y_min)
					j_max = N-1; 
				else //if (alpha_min != alpha_y_min)
					j_max =  floor(( alpha_min*(vertex_x2_y - vertex_x1_y) + vertex_x1_y - boundary_voxel_y )*inv_volumn_y) ;
				
				if (alpha_max == alpha_y_max)
					j_min = 0; 
				else //if (alpha_max != alpha_y_max)
					j_min =  floor(( alpha_max*(vertex_x2_y - vertex_x1_y) + vertex_x1_y - boundary_voxel_y )*inv_volumn_y) + 1 ;

			}	
            // Note: overall, j_min is the most bottom y-plane, j_max the most top y-plane,
            // and the initial point (the first interacted position on the boundary) NOT included.
            
			//Z direction 
            if (fabs(vertex_x1_z-vertex_x2_z)<volumn_z*1e-6 )  
            {
				k_min =  floor(( vertex_x1_z - boundary_voxel_z )*inv_volumn_z) + 1 ;                
				k_max =  floor(( vertex_x1_z - boundary_voxel_z )*inv_volumn_z) ;    
                // Note: this condition can be combined into either of the two branches.
            }   
            else if (vertex_x1_z < vertex_x2_z)
			{	
				if (alpha_min == alpha_z_min)
					k_min = 1; 
				else //if (alpha_min != alpha_z_min)
					k_min =  floor(( alpha_min*(vertex_x2_z - vertex_x1_z) + vertex_x1_z  - boundary_voxel_z )*inv_volumn_z) + 1 ;
				
				if (alpha_max == alpha_z_max)
					k_max = ZETA; 
				else //if (alpha_max != alpha_z_max)
					k_max =  floor(( alpha_max*(vertex_x2_z - vertex_x1_z) + vertex_x1_z  - boundary_voxel_z )*inv_volumn_z) ;

			}	
			else //if (vertex_x1_z > vertex_x2_z)
			{	
				if (alpha_min == alpha_z_min)
					k_max = ZETA-1; 
				else //if (alpha_min != alpha_z_min)
					k_max =  floor(( alpha_min*(vertex_x2_z - vertex_x1_z) + vertex_x1_z  - boundary_voxel_z )*inv_volumn_z) ;
				
				if (alpha_max == alpha_z_max)
					k_min = 0; 
				else //if (alpha_max != alpha_z_max)
					k_min =  floor(( alpha_max*(vertex_x2_z - vertex_x1_z) + vertex_x1_z  -boundary_voxel_z )*inv_volumn_z) + 1 ;

			}	
            

        /************ initialization (i,j,k) (alpha_x_1,alpha_y_1,alpha_z_1)**************************/            
        // Note: (i,j,k) is the current x,y,z plane index (@ the initial point at the boundary)
        // Note: (alpha_x,alpha_y,alpha_z) is the next x,y,z plane to go.
            
		N_total_sec = i_max - i_min + 1 + j_max - j_min +1 + k_max-k_min +1; 
            // i.e. N_p (25)

		if (fabs(vertex_x1_x-vertex_x2_x)<volumn_x*1e-6 )  
        {
            alpha_x = MAX_infi;
            i = i_min-1;
        }
        else if (vertex_x1_x < vertex_x2_x)
        {
 			alpha_x = (volumn_x * i_min + boundary_voxel_x - vertex_x1_x )* inv_x_diff;
            i = i_min - 1;   
        }        
		else if (vertex_x1_x > vertex_x2_x) 			
        {
            alpha_x = (volumn_x * i_max + boundary_voxel_x - vertex_x1_x )* inv_x_diff;
            i = i_max + 1;
        }
            // Note: alpha_x_1 is the intersection where the path hit the 1st x plane inside the recon region
		
		if (fabs(vertex_x1_y-vertex_x2_y)<volumn_y*1e-6 )  
        {
            alpha_y = MAX_infi;
            j = j_min-1;
        }
        else 
            if (vertex_x1_y < vertex_x2_y)
        {            
 			alpha_y = (volumn_y * j_min + boundary_voxel_y - vertex_x1_y )* inv_y_diff;
            j = j_min - 1;
        }
		else if (vertex_x1_y >= vertex_x2_y)
        {
 			alpha_y = (volumn_y * j_max + boundary_voxel_y - vertex_x1_y )* inv_y_diff;
            j = j_max + 1;
        }            
            // Note: alpha_y_1 is the intersection where the path hit the 1st y plane inside the recon region
	
		if (fabs(vertex_x1_z-vertex_x2_z)<volumn_z*1e-6 )  
        {
            alpha_z = MAX_infi;
            k = k_min-1;
        }
        else if (vertex_x1_z <= vertex_x2_z)
        {
 			alpha_z = (volumn_z * k_min + boundary_voxel_z - vertex_x1_z )* inv_z_diff;
            k = k_min - 1;
        }            
		else if (vertex_x1_z > vertex_x2_z)
        {
            alpha_z = (volumn_z * k_max + boundary_voxel_z - vertex_x1_z )* inv_z_diff;
            k = k_max + 1;
        }        
                
                                     
        /************ initialization (voxel_i,voxel_j,voxel_k) **************************/            
        // Note: (voxel_i,voxel_j,voxel_k) is the current x,y,z voxel index (@ the initial point at the boundary)
        
        if (vertex_x1_x < vertex_x2_x)
            voxel_i = i_min-1;
        else 
            voxel_i = i_max;
                
        if (vertex_x1_y < vertex_x2_y)
            voxel_j = j_min-1;
        else 
            voxel_j = j_max;
               
        if (vertex_x1_z < vertex_x2_z)
            voxel_k = k_min-1;
        else 
            voxel_k = k_max;                              
        
        /***************** Updating alpha_x, alpha_y, alpha_z, ************************/
        
        // Note: (alpha_x, alpha_y, alpha_z) the intersection where the path hit the next (i.e. 1st here ) x/y/z plane inside the recon
        
        d_x1_x2 = sqrt((vertex_x2_x-vertex_x1_x)*(vertex_x2_x-vertex_x1_x) + (vertex_x2_y-vertex_x1_y)*(vertex_x2_y - vertex_x1_y) + (vertex_x2_z-vertex_x1_z)*(vertex_x2_z-vertex_x1_z) );
      	                
        alpha_c = alpha_min;    // intersection where the path hit the 1st plane at the boundary of recon region

        // Note : (i,j,k) is the (x,y,z) plane index of the current intersection (with a certain plane)
        // If i or j or k should not be an integer, then its predecessor (along the ray)
        
        while (alpha_max - alpha_c > 1e-16)
       	{
            
          if ((voxel_i > M-1)||(voxel_i <0) || (voxel_j > N-1)||(voxel_j <0) || (voxel_k > ZETA-1)||(voxel_k <0))
          {
                alpha_c = alpha_max +1;  // to terminate the loop
          }         
          else
          {
		
  			if ( (alpha_x < alpha_y) && (alpha_x < alpha_z))
                            // alpha_x is the nearest, so update alpha_x
            {
				one_ray_length += d_x1_x2 * (alpha_x - alpha_c);  //(30)
                one_ray_sum += d_x1_x2 * (alpha_x - alpha_c) * d_f[voxel_k*M*N + voxel_j*M + voxel_i];
                                                                //(31)		              
				alpha_c = alpha_x;          //(33)   Update the current location
				N_total_sec = N_total_sec - 1; 
                
                if (vertex_x1_x < vertex_x2_x)
                {
					i++;
                    voxel_i++;
                    next_alpha_index = i+1;
                }
                if (vertex_x1_x > vertex_x2_x)
                {
                    i--;      //(29)   
                    voxel_i--;
                    next_alpha_index = i-1;
                }
                alpha_x = (volumn_x * next_alpha_index + boundary_voxel_x - vertex_x1_x )* inv_x_diff;
                
           	}
            
			else if ( (alpha_y < alpha_x) && (alpha_y < alpha_z) )
                            // alpha_y is the nearest, so update alpha_y
           	{                        
				one_ray_length += d_x1_x2 * (alpha_y - alpha_c);
                one_ray_sum += d_x1_x2 * (alpha_y - alpha_c) * d_f[voxel_k*M*N + voxel_j*M + voxel_i];
				
                alpha_c = alpha_y; 
                N_total_sec = N_total_sec -1;                
                
                if (vertex_x1_y < vertex_x2_y)
                {
					j++;
                    voxel_j++;
                    next_alpha_index = j+1;
                }
                else if (vertex_x1_y > vertex_x2_y)
                {
					j--;
                    voxel_j--;
                    next_alpha_index = j-1;
                }   
                alpha_y = (volumn_y * next_alpha_index + boundary_voxel_y - vertex_x1_y )* inv_y_diff;                
           	}
            
			else if ( (alpha_z < alpha_x) && (alpha_z < alpha_y) )
                        // alpha_z is the nearest, so update alpha_z                
            {				
				one_ray_length += d_x1_x2 * (alpha_z - alpha_c);
                one_ray_sum += d_x1_x2 * (alpha_z - alpha_c) * d_f[voxel_k*M*N + voxel_j*M + voxel_i];
				
                alpha_c = alpha_z; 
                N_total_sec = N_total_sec -1;                
                
                if (vertex_x1_z < vertex_x2_z)
                {
					k++;
                    voxel_k++;
                    next_alpha_index = k+1;
                }
                else if (vertex_x1_z > vertex_x2_z)
                {
					k--;
                    voxel_k--;
                    next_alpha_index = k-1;
                }
                alpha_z = (volumn_z * next_alpha_index + boundary_voxel_z - vertex_x1_z )* inv_z_diff;
                
            }
		     
			else if ( (alpha_x == alpha_y) && (alpha_x < alpha_z) )
                        //x = y < z
            {        

				one_ray_length += d_x1_x2 * (alpha_x - alpha_c);  //(30)
                one_ray_sum += d_x1_x2 * (alpha_x - alpha_c) * d_f[voxel_k*M*N + voxel_j*M + voxel_i];  //(31)	
                                                                	              
				alpha_c = alpha_x;          //(33)   Update the current location
				N_total_sec = N_total_sec - 2; 
                                  
                if (vertex_x1_x < vertex_x2_x)
                {
					i = i + 1;
                    voxel_i = voxel_i +1;
                    next_alpha_index = i+1;
                }
                if (vertex_x1_x > vertex_x2_x)
                {
                    i = i - 1;      //(29)   
                    voxel_i = voxel_i-1;
                    next_alpha_index = i-1;                                        
                }
                alpha_x = (volumn_x * next_alpha_index + boundary_voxel_x - vertex_x1_x )* inv_x_diff;
                
                if (vertex_x1_y < vertex_x2_y)
                {
					j = j + 1;
                    voxel_j = voxel_j+1;
                    next_alpha_index = j+1; 
                }
                else if (vertex_x1_y > vertex_x2_y)
                {
					j = j - 1;
                    voxel_j = voxel_j-1;
                    next_alpha_index = j-1;
                }                
                alpha_y = (volumn_y * next_alpha_index + boundary_voxel_y - vertex_x1_y )* inv_y_diff;
                
            }
            
          	else if ( (alpha_x == alpha_z) && (alpha_x < alpha_y))// && (sphere_range<=1.0f) )
                        // x = z < y;
            {                      
				one_ray_length += d_x1_x2 * (alpha_x - alpha_c);  //(30)
                one_ray_sum += d_x1_x2 * (alpha_x - alpha_c) * d_f[voxel_k*M*N + voxel_j*M + voxel_i];  //(31)	
                                                                	              
				alpha_c = alpha_x;          //(33)   Update the current location
				N_total_sec = N_total_sec - 2; 

                if (vertex_x1_x < vertex_x2_x)
                {
					i = i + 1;
                    voxel_i = voxel_i +1;
                    next_alpha_index = i+1;                    
                }
                if (vertex_x1_x > vertex_x2_x)
                {
                    i = i - 1;      //(29)   
                    voxel_i = voxel_i-1;
                    next_alpha_index = i-1;                 
                }
                alpha_x = (volumn_x * next_alpha_index + boundary_voxel_x - vertex_x1_x )* inv_x_diff;
                
                if (vertex_x1_z < vertex_x2_z)
                {
					k = k + 1;
                    voxel_k = voxel_k+1;
                    next_alpha_index = k+1;
                }
                else if (vertex_x1_z > vertex_x2_z)
                {
					k = k - 1;
                    voxel_k = voxel_k-1;
                    next_alpha_index = k-1;
                }                
                alpha_z = (volumn_z * next_alpha_index + boundary_voxel_z - vertex_x1_z )* inv_z_diff;

            }
            
			else if ( (alpha_y == alpha_z) && (alpha_y < alpha_x))// && (sphere_range<=1.0f) )
                      	// y = z < x        	
            {            	
				one_ray_length += d_x1_x2 * (alpha_y - alpha_c);
                one_ray_sum += d_x1_x2 * (alpha_y - alpha_c) * d_f[voxel_k*M*N + voxel_j*M + voxel_i];
				
                alpha_c = alpha_y; 
                N_total_sec = N_total_sec -2;                
                
                if (vertex_x1_y < vertex_x2_y)
                {
					j = j + 1;
                    voxel_j = voxel_j+1;
                    next_alpha_index = j+1;
                }
                else if (vertex_x1_y > vertex_x2_y)
                {
					j = j - 1;
                    voxel_j = voxel_j-1;
                    next_alpha_index = j-1;
                }   
                alpha_y = (volumn_y * next_alpha_index + boundary_voxel_y - vertex_x1_y )* inv_y_diff;
                
                if (vertex_x1_z < vertex_x2_z)
                {
					k = k + 1;
                    voxel_k = voxel_k+1;
                    next_alpha_index = k+1;
                }
                else if (vertex_x1_z > vertex_x2_z)
                {
					k = k - 1;
                    voxel_k = voxel_k-1;
                    next_alpha_index = k-1;
                }  
                alpha_z = (volumn_z * next_alpha_index + boundary_voxel_z - vertex_x1_z )* inv_z_diff;
                
            }
			
          	else if ( (alpha_x == alpha_z) && (alpha_x == alpha_y))// && (sphere_range<=1.0f) )
                        // x=y=z            
            {
				one_ray_length += d_x1_x2 * (alpha_x - alpha_c);  //(30)
                one_ray_sum += d_x1_x2 * (alpha_x - alpha_c) * d_f[voxel_k*M*N + voxel_j*M + voxel_i];  //(31)	
                                                                	              
				alpha_c = alpha_x;          //(33)   Update the current location
				N_total_sec = N_total_sec - 3; 

                if (vertex_x1_x < vertex_x2_x)
                {
					i = i + 1;
                    voxel_i = voxel_i +1;
                    next_alpha_index = i+1;
                }
                if (vertex_x1_x > vertex_x2_x)
                {
                    i = i - 1;      //(29)   
                    voxel_i = voxel_i-1;
                    next_alpha_index = i-1;                   
                }
                alpha_x = (volumn_x * next_alpha_index + boundary_voxel_x - vertex_x1_x )* inv_x_diff;
                
                if (vertex_x1_y < vertex_x2_y)
                {
					j = j + 1;
                    voxel_j = voxel_j+1;
                    next_alpha_index = j+1;
                }
                else if (vertex_x1_y > vertex_x2_y)
                {
					j = j - 1;
                    voxel_j = voxel_j-1;
                    next_alpha_index = j-1;
                }   
                alpha_y = (volumn_y * next_alpha_index + boundary_voxel_y - vertex_x1_y )* inv_y_diff;
                
                if (vertex_x1_z < vertex_x2_z)
                {
					k = k + 1;
                    voxel_k = voxel_k+1;
                    next_alpha_index = k+1;
                }
                else if (vertex_x1_z > vertex_x2_z)
                {
					k = k - 1;
                    voxel_k = voxel_k-1;
                    next_alpha_index = k-1;
                }  
                alpha_z = (volumn_z * next_alpha_index + boundary_voxel_z - vertex_x1_z )* inv_z_diff;
           	}
          }
       	}// end tracing the ray                      
     }//end if the ray interacts with the volume
    
    if (one_ray_length < volumn_z*1e-6)            
        d_proj_correction[proj_pixel_index] = 0.0;
	else
    {
    	if (command == 0)
        	d_proj_correction[proj_pixel_index] = one_ray_sum; // forward operator
            
         else if (command == 1)                
         	d_proj_correction[proj_pixel_index] = (d_proj_data[proj_pixel_index] - one_ray_sum)/one_ray_length;                                                                     // projection correction (for SART)
	}    
    
//    __syncthreads();
    
}


__global__ void forward_ray_driven_3d_kernel_correction_separate(float *d_f , float *d_proj_sumLen, float *d_proj_weightedLen, float sin_theta, float cos_theta, int subVolIdx)

{
	// d_f: 3D object array;    d_f[i,j,k] = d_f [k*M*N+j*M+i]; 
    // d_proj_data: 2D projection acquired at the angle of t_theta
	// d_proj_sumLen: 2D projection correction,  (output of this function. i.e. c(i) in the paper)    
        
    int Detector_x_idx = threadIdx.x + blockDim.x * blockIdx.x;
    int Detector_z_idx = blockIdx.y;
    
    int proj_pixel_index = Detector_z_idx * R + Detector_x_idx;               
    
    // Source positions (X2): Coordinate in (x,y,z) system --- 
    float vertex_x2_x,vertex_x2_y,vertex_x2_z;    
    if (CT_style==0)   //CBCT
    {
        vertex_x2_x = Source_x * cos_theta - Source_y * sin_theta;
        vertex_x2_y = Source_x * sin_theta + Source_y * cos_theta;
        vertex_x2_z = Source_z;
    }
    else if (CT_style==1) //FBCT
    {
        vertex_x2_x = Source_x * cos_theta - Source_y * sin_theta;
        vertex_x2_y = Source_x * sin_theta + Source_y * cos_theta;
        vertex_x2_z = Detector_Zmin + Detector_z_idx * Detector_pixel_x;        
    }    
    else if (CT_style==2) //parallel beam
    {
        vertex_x2_x = Source_x * cos_theta - (Detector_Ymin +  Detector_x_idx * Detector_pixel_x) * sin_theta;
        vertex_x2_y = Source_x * sin_theta + (Detector_Ymin +  Detector_x_idx * Detector_pixel_x) * cos_theta;
        vertex_x2_z = Detector_Zmin + Detector_z_idx * Detector_pixel_x; 
    }
        
    // Detector element center positions (X1): Coordinate in (x,y,z) system --- 
    float vertex_x1_x,vertex_x1_y,vertex_x1_z;
    vertex_x1_x = DOD * cos_theta - (Detector_Ymin +  Detector_x_idx * Detector_pixel_x) * sin_theta;
    vertex_x1_y = DOD * sin_theta + (Detector_Ymin +  Detector_x_idx * Detector_pixel_x) * cos_theta;
    vertex_x1_z = Detector_Zmin + Detector_z_idx * Detector_pixel_x;        
    
        //  Notice: in this system, vertex_x1_x < 0 < vertex_x2_x    
            
    float inv_x_diff = 1.0f / (vertex_x2_x - vertex_x1_x);
    float inv_y_diff = 1.0f / (vertex_x2_y - vertex_x1_y);
    float inv_z_diff = 1.0f / (vertex_x2_z - vertex_x1_z);    
    
    float BOUNDARY_VOXEL_Z = boundary_voxel_z + volumn_z*ZETA/Number_of_Devices*subVolIdx;
    int ZETA_new = ZETA/Number_of_Devices;
    
    /*****************************************/
                
    float alpha_x_min= 0.0f, alpha_y_min= 0.0f, alpha_z_min= 0.0f;
    float alpha_x_max= 0.0f, alpha_y_max= 0.0f, alpha_z_max= 0.0f;
    float alpha_min= 0.0f, alpha_max= 0.0f;
        
	int i_min=0, j_min=0, k_min=0;
	int i_max=0, j_max=0, k_max=0;
    int i=0, j=0, k=0;
    int voxel_i=0, voxel_j=0, voxel_k=0;
    
	float alpha_x=0.0f, alpha_y=0.0f, alpha_z=0.0f;  
    float one_ray_sum = 0.0f;
    float one_ray_length = 0.0f; 

    float alpha_c= 0.0f;
    float d_x1_x2= 0.0f;

	int N_total_sec=0; 
    
    int next_alpha_index;
            
	/**** Step 1 :find out alpha_min, alpha_max ********/
    
    if ( (CT_style==1 || CT_style==2) && (Detector_z_idx<Z_prj/Number_of_Devices*subVolIdx || Detector_z_idx>=Z_prj/Number_of_Devices*(subVolIdx+1)) )
    {
        one_ray_sum = 0.0f; 
        one_ray_length = 0.00f; 
    }    

    else //  if ( (vertex_x1_x != vertex_x2_x) && (vertex_x1_y != vertex_x2_y) )
    {

        alpha_min = (boundary_voxel_x + volumn_x*0 - vertex_x1_x )* inv_x_diff; //(9)
        alpha_max = (boundary_voxel_x + volumn_x*M - vertex_x1_x )* inv_x_diff;
        // Notice: it is still unsure here which one is the parametric value of the first intersection point of the ray with the x-plane
        // It depends on whether source or detector lies on the left side of the reconstruction region at this time

        alpha_x_min = fmin(alpha_min, alpha_max);   //(5)
        alpha_x_max = fmax(alpha_min, alpha_max );  //(6) 
                
        alpha_min = (boundary_voxel_y + volumn_y*0 - vertex_x1_y )* inv_y_diff;
        alpha_max = (boundary_voxel_y + volumn_y*N - vertex_x1_y )* inv_y_diff;

        alpha_y_min = fmin(alpha_min, alpha_max);   //(7)
        alpha_y_max = fmax(alpha_min, alpha_max );  //(8)
        
        alpha_min = (BOUNDARY_VOXEL_Z + volumn_z*0 - vertex_x1_z )* inv_z_diff;
        alpha_max = (BOUNDARY_VOXEL_Z + volumn_z*ZETA_new - vertex_x1_z )* inv_z_diff;        
        
        alpha_z_min = fmin(alpha_min, alpha_max);   
        alpha_z_max = fmax(alpha_min, alpha_max );  

        // alpha_min / alpha_max reused 
        alpha_min = fmax(fmax(alpha_x_min, alpha_y_min), fmax(alpha_y_min, alpha_z_min)); //(3)
        // i.e. alpha_min = fmax(alpha_x_min,alpha_y_min,alpha_z_min)
        // it indicates the point where the path interacts with the near boundary of reconstruction region        

        alpha_max = fmin(fmin(alpha_x_max, alpha_y_max), fmin(alpha_y_max, alpha_z_max)); //(4)
        // i.e. alpha_max = fmin(alpha_x_max,alpha_y_max,alpha_z_max)
        // it indicates the point where the path last interacts with the far boundary of reconstruction region        
        
        /********Step 2,3: Find i_max, i_min***************/
        
     if (alpha_max <= alpha_min)   // It means no interaction of the ray and the volume
     {
        one_ray_length = 0.0f ;
        one_ray_sum=0.0f;  
     }
	 else 
     {
			// X direction 
			if (vertex_x1_x < vertex_x2_x)
			{	
				if (alpha_min == alpha_x_min)
					i_min = 1;      //(11)
				else //if (alpha_min != alpha_x_min)
					i_min =  floor(( alpha_min*(vertex_x2_x - vertex_x1_x) + vertex_x1_x - boundary_voxel_x)*inv_volumn_x) + 1 ;
                                    //(12)
                     /* Note: i_min is the index of the 1st x plane where the path interacts inside the reconstruction region
                      * It is not the index of alpha_x_min
                      */                
				if (alpha_max == alpha_x_max)
					i_max = M;      //(13)
				else //if (alpha_max != alpha_x_max)
					i_max =  floor(( alpha_max*(vertex_x2_x - vertex_x1_x) + vertex_x1_x - boundary_voxel_x)*inv_volumn_x) ;
                                    //(14)
                     // Note: i_max is the index of the last x plane where the path interacts with the reconstruction region (inside or boundary)                      
			}	
			else //if (vertex_x1_x >= vertex_x2_x)
			{	
				if (alpha_min == alpha_x_min)
					i_max = M-1;    //(15)
				else //if (alpha_min != alpha_x_min)
					i_max =  floor(( alpha_min*(vertex_x2_x - vertex_x1_x) + vertex_x1_x - boundary_voxel_x)*inv_volumn_x) ;				
                                    //(16)
				if (alpha_max == alpha_x_max)
					i_min = 0;      //(17)
				else //if (alpha_max != alpha_x_max)
					i_min =  floor(( alpha_max*(vertex_x2_x - vertex_x1_x) + vertex_x1_x - boundary_voxel_x)*inv_volumn_x) + 1 ;
                                    //(18)
			}	
            // Note: overall, i_min is the most left x-plane, i_max the most right x-plane,
            // and the initial point (the first interacted position on the boundary) NOT included.            
               
			//Y direction 
			if (vertex_x1_y < vertex_x2_y)
			{	
				if (alpha_min == alpha_y_min)
					j_min = 1; 
				else //f (alpha_min != alpha_y_min)
					j_min =  floor(( alpha_min*(vertex_x2_y - vertex_x1_y) + vertex_x1_y - boundary_voxel_y)*inv_volumn_y) + 1 ;
				
				if (alpha_max == alpha_y_max)
					j_max = N; 
				else //if (alpha_max != alpha_y_max)
					j_max =  floor(( alpha_max*(vertex_x2_y - vertex_x1_y) + vertex_x1_y - boundary_voxel_y)*inv_volumn_y) ;

			}	
			else //if (vertex_x1_y >= vertex_x2_y)
			{	
				if (alpha_min == alpha_y_min)
					j_max = N-1; 
				else //if (alpha_min != alpha_y_min)
					j_max =  floor(( alpha_min*(vertex_x2_y - vertex_x1_y) + vertex_x1_y - boundary_voxel_y )*inv_volumn_y) ;
				
				if (alpha_max == alpha_y_max)
					j_min = 0; 
				else //if (alpha_max != alpha_y_max)
					j_min =  floor(( alpha_max*(vertex_x2_y - vertex_x1_y) + vertex_x1_y - boundary_voxel_y )*inv_volumn_y) + 1 ;

			}	
            // Note: overall, j_min is the most bottom y-plane, j_max the most top y-plane,
            // and the initial point (the first interacted position on the boundary) NOT included.
            
			//Z direction 
            if (vertex_x1_z < vertex_x2_z)
			{	
				if (alpha_min == alpha_z_min)
					k_min = 1; 
				else //if (alpha_min != alpha_z_min)
					k_min =  floor(( alpha_min*(vertex_x2_z - vertex_x1_z) + vertex_x1_z  - BOUNDARY_VOXEL_Z )*inv_volumn_z) + 1 ;
				
				if (alpha_max == alpha_z_max)
					k_max = ZETA_new; 
				else //if (alpha_max != alpha_z_max)
					k_max =  floor(( alpha_max*(vertex_x2_z - vertex_x1_z) + vertex_x1_z  - BOUNDARY_VOXEL_Z )*inv_volumn_z) ;

			}	
			else //if (vertex_x1_z >= vertex_x2_z)
			{	
				if (alpha_min == alpha_z_min)
					k_max = ZETA_new-1; 
				else //if (alpha_min != alpha_z_min)
					k_max =  floor(( alpha_min*(vertex_x2_z - vertex_x1_z) + vertex_x1_z  - BOUNDARY_VOXEL_Z )*inv_volumn_z) ;
				
				if (alpha_max == alpha_z_max)
					k_min = 0; 
				else //if (alpha_max != alpha_z_max)
					k_min =  floor(( alpha_max*(vertex_x2_z - vertex_x1_z) + vertex_x1_z  -BOUNDARY_VOXEL_Z )*inv_volumn_z) + 1 ;

			}	
            
        /************ initialization (i,j,k) (alpha_x_1,alpha_y_1,alpha_z_1)**************************/            
        // Note: (i,j,k) is the current x,y,z plane index (@ the initial point at the boundary)
        // Note: (alpha_x,alpha_y,alpha_z) is the next x,y,z plane to go.
            
		N_total_sec = i_max - i_min + 1 + j_max - j_min +1 + k_max-k_min +1; 
            // i.e. N_p (25)

        if (fabs(vertex_x1_x-vertex_x2_x)<volumn_x*1e-6 )  
        {
            alpha_x = MAX_infi;
            i = i_min-1;
        }
        else if (vertex_x1_x < vertex_x2_x)
        {
 			alpha_x = (volumn_x * i_min + boundary_voxel_x - vertex_x1_x )* inv_x_diff;
            i = i_min - 1;   
        }        
		else if (vertex_x1_x > vertex_x2_x) 			
        {
            alpha_x = (volumn_x * i_max + boundary_voxel_x - vertex_x1_x )* inv_x_diff;
            i = i_max + 1;
        }
            // Note: alpha_x_1 is the intersection where the path hit the 1st x plane inside the recon region
		
		if (fabs(vertex_x1_y-vertex_x2_y)<volumn_y*1e-6 )  
        {
            alpha_y = MAX_infi;
            j = j_min-1;
        }
        else if (vertex_x1_y < vertex_x2_y)
        {            
 			alpha_y = (volumn_y * j_min + boundary_voxel_y - vertex_x1_y )* inv_y_diff;
            j = j_min - 1;
        }
		else if (vertex_x1_y > vertex_x2_y)
        {
 			alpha_y = (volumn_y * j_max + boundary_voxel_y - vertex_x1_y )* inv_y_diff;
            j = j_max + 1;
        }            
            // Note: alpha_y_1 is the intersection where the path hit the 1st y plane inside the recon region
	
		if (fabs(vertex_x1_z-vertex_x2_z)<volumn_z*1e-6 )  
        {
            alpha_z = MAX_infi;
            k = k_min-1;
        }
        else if (vertex_x1_z < vertex_x2_z)
        {
 			alpha_z = (volumn_z * k_min + BOUNDARY_VOXEL_Z - vertex_x1_z )* inv_z_diff;
            k = k_min - 1;
        }            
		else if (vertex_x1_z > vertex_x2_z)
        {
            alpha_z = (volumn_z * k_max + BOUNDARY_VOXEL_Z - vertex_x1_z )* inv_z_diff;
            k = k_max + 1;
        }        
                                     
        /************ initialization (voxel_i,voxel_j,voxel_k) **************************/            
        // Note: (voxel_i,voxel_j,voxel_k) is the current x,y,z voxel index (@ the initial point at the boundary)
        
        if (vertex_x1_x < vertex_x2_x)
            voxel_i = i_min-1;
        else 
            voxel_i = i_max;
                
        if (vertex_x1_y < vertex_x2_y)
            voxel_j = j_min-1;
        else 
            voxel_j = j_max;
               
        if (vertex_x1_z < vertex_x2_z)
            voxel_k = k_min-1;
        else 
            voxel_k = k_max;                              
        
        /***************** Updating alpha_x, alpha_y, alpha_z, ************************/
        
        // Note: (alpha_x, alpha_y, alpha_z) the intersection where the path hit the next (i.e. 1st here ) x/y/z plane inside the recon
        
        d_x1_x2 = sqrt((vertex_x2_x-vertex_x1_x)*(vertex_x2_x-vertex_x1_x) + (vertex_x2_y-vertex_x1_y)*(vertex_x2_y - vertex_x1_y) + (vertex_x2_z-vertex_x1_z)*(vertex_x2_z-vertex_x1_z) );
      	                
        alpha_c = alpha_min;    // intersection where the path hit the 1st plane at the boundary of recon region

        // Note : (i,j,k) is the (x,y,z) plane index of the current intersection (with a certain plane)
        // If i or j or k should not be an integer, then its predecessor (along the ray)
        
        while (alpha_max - alpha_c > 1e-16)
       	{
            
          if ((voxel_i > M-1)||(voxel_i <0) || (voxel_j > N-1)||(voxel_j <0) || (voxel_k > ZETA_new-1)||(voxel_k <0))
          {
                alpha_c = alpha_max +1;  // to terminate the loop
          }         
          else
          {
		
  			if ( (alpha_x < alpha_y) && (alpha_x < alpha_z))
                            // alpha_x is the nearest, so update alpha_x
            {
				one_ray_length += d_x1_x2 * (alpha_x - alpha_c);  //(30)
                one_ray_sum += d_x1_x2 * (alpha_x - alpha_c) * d_f[voxel_k*M*N + voxel_j*M + voxel_i];
                                                                //(31)		              
				alpha_c = alpha_x;          //(33)   Update the current location
				N_total_sec = N_total_sec - 1; 
                
                if (vertex_x1_x < vertex_x2_x)
                {
					i++;
                    voxel_i++;
                    next_alpha_index = i+1;
                }
                if (vertex_x1_x > vertex_x2_x)
                {
                    i--;      //(29)   
                    voxel_i--;
                    next_alpha_index = i-1;
                }
                alpha_x = (volumn_x * next_alpha_index + boundary_voxel_x - vertex_x1_x )* inv_x_diff;
                
           	}
            
			else if ( (alpha_y < alpha_x) && (alpha_y < alpha_z) )
                            // alpha_y is the nearest, so update alpha_y
           	{                        
				one_ray_length += d_x1_x2 * (alpha_y - alpha_c);
                one_ray_sum += d_x1_x2 * (alpha_y - alpha_c) * d_f[voxel_k*M*N + voxel_j*M + voxel_i];
				
                alpha_c = alpha_y; 
                N_total_sec = N_total_sec -1;                
                
                if (vertex_x1_y < vertex_x2_y)
                {
					j++;
                    voxel_j++;
                    next_alpha_index = j+1;
                }
                else if (vertex_x1_y > vertex_x2_y)
                {
					j--;
                    voxel_j--;
                    next_alpha_index = j-1;
                }   
                alpha_y = (volumn_y * next_alpha_index + boundary_voxel_y - vertex_x1_y )* inv_y_diff;                
           	}
            
			else if ( (alpha_z < alpha_x) && (alpha_z < alpha_y) )
                        // alpha_z is the nearest, so update alpha_z                
            {				
				one_ray_length += d_x1_x2 * (alpha_z - alpha_c);
                one_ray_sum += d_x1_x2 * (alpha_z - alpha_c) * d_f[voxel_k*M*N + voxel_j*M + voxel_i];
				
                alpha_c = alpha_z; 
                N_total_sec = N_total_sec -1;                
                
                if (vertex_x1_z < vertex_x2_z)
                {
					k++;
                    voxel_k++;
                    next_alpha_index = k+1;
                }
                else if (vertex_x1_z > vertex_x2_z)
                {
					k--;
                    voxel_k--;
                    next_alpha_index = k-1;
                }
                alpha_z = (volumn_z * next_alpha_index + BOUNDARY_VOXEL_Z - vertex_x1_z )* inv_z_diff;
                
            }
		     
			else if ( (alpha_x == alpha_y) && (alpha_x < alpha_z) )
                        //x = y < z
            {        

				one_ray_length += d_x1_x2 * (alpha_x - alpha_c);  //(30)
                one_ray_sum += d_x1_x2 * (alpha_x - alpha_c) * d_f[voxel_k*M*N + voxel_j*M + voxel_i];  //(31)	
                                                                	              
				alpha_c = alpha_x;          //(33)   Update the current location
				N_total_sec = N_total_sec - 2; 
                                  
                if (vertex_x1_x < vertex_x2_x)
                {
					i = i + 1;
                    voxel_i = voxel_i +1;
                    next_alpha_index = i+1;
                }
                if (vertex_x1_x > vertex_x2_x)
                {
                    i = i - 1;      //(29)   
                    voxel_i = voxel_i-1;
                    next_alpha_index = i-1;                                        
                }
                alpha_x = (volumn_x * next_alpha_index + boundary_voxel_x - vertex_x1_x )* inv_x_diff;
                
                if (vertex_x1_y < vertex_x2_y)
                {
					j = j + 1;
                    voxel_j = voxel_j+1;
                    next_alpha_index = j+1; 
                }
                else if (vertex_x1_y > vertex_x2_y)
                {
					j = j - 1;
                    voxel_j = voxel_j-1;
                    next_alpha_index = j-1;
                }                
                alpha_y = (volumn_y * next_alpha_index + boundary_voxel_y - vertex_x1_y )* inv_y_diff;
                
            }
            
          	else if ( (alpha_x == alpha_z) && (alpha_x < alpha_y))// && (sphere_range<=1.0f) )
                        // x = z < y;
            {                      
				one_ray_length += d_x1_x2 * (alpha_x - alpha_c);  //(30)
                one_ray_sum += d_x1_x2 * (alpha_x - alpha_c) * d_f[voxel_k*M*N + voxel_j*M + voxel_i];  //(31)	
                                                                	              
				alpha_c = alpha_x;          //(33)   Update the current location
				N_total_sec = N_total_sec - 2; 

                if (vertex_x1_x < vertex_x2_x)
                {
					i = i + 1;
                    voxel_i = voxel_i +1;
                    next_alpha_index = i+1;                    
                }
                if (vertex_x1_x > vertex_x2_x)
                {
                    i = i - 1;      //(29)   
                    voxel_i = voxel_i-1;
                    next_alpha_index = i-1;                 
                }
                alpha_x = (volumn_x * next_alpha_index + boundary_voxel_x - vertex_x1_x )* inv_x_diff;
                
                if (vertex_x1_z < vertex_x2_z)
                {
					k = k + 1;
                    voxel_k = voxel_k+1;
                    next_alpha_index = k+1;
                }
                else if (vertex_x1_z > vertex_x2_z)
                {
					k = k - 1;
                    voxel_k = voxel_k-1;
                    next_alpha_index = k-1;
                }                
                alpha_z = (volumn_z * next_alpha_index + BOUNDARY_VOXEL_Z - vertex_x1_z )* inv_z_diff;

            }
            
			else if ( (alpha_y == alpha_z) && (alpha_y < alpha_x))// && (sphere_range<=1.0f) )
                      	// y = z < x        	
            {            	
				one_ray_length += d_x1_x2 * (alpha_y - alpha_c);
                one_ray_sum += d_x1_x2 * (alpha_y - alpha_c) * d_f[voxel_k*M*N + voxel_j*M + voxel_i];
				
                alpha_c = alpha_y; 
                N_total_sec = N_total_sec -2;                
                
                if (vertex_x1_y < vertex_x2_y)
                {
					j = j + 1;
                    voxel_j = voxel_j+1;
                    next_alpha_index = j+1;
                }
                else if (vertex_x1_y > vertex_x2_y)
                {
					j = j - 1;
                    voxel_j = voxel_j-1;
                    next_alpha_index = j-1;
                }   
                alpha_y = (volumn_y * next_alpha_index + boundary_voxel_y - vertex_x1_y )* inv_y_diff;
                
                if (vertex_x1_z < vertex_x2_z)
                {
					k = k + 1;
                    voxel_k = voxel_k+1;
                    next_alpha_index = k+1;
                }
                else if (vertex_x1_z > vertex_x2_z)
                {
					k = k - 1;
                    voxel_k = voxel_k-1;
                    next_alpha_index = k-1;
                }  
                alpha_z = (volumn_z * next_alpha_index + BOUNDARY_VOXEL_Z - vertex_x1_z )* inv_z_diff;
                
            }
			
          	else if ( (alpha_x == alpha_z) && (alpha_x == alpha_y))// && (sphere_range<=1.0f) )
                        // x=y=z            
            {
				one_ray_length += d_x1_x2 * (alpha_x - alpha_c);  //(30)
                one_ray_sum += d_x1_x2 * (alpha_x - alpha_c) * d_f[voxel_k*M*N + voxel_j*M + voxel_i];  //(31)	
                                                                	              
				alpha_c = alpha_x;          //(33)   Update the current location
				N_total_sec = N_total_sec - 3; 

                if (vertex_x1_x < vertex_x2_x)
                {
					i = i + 1;
                    voxel_i = voxel_i +1;
                    next_alpha_index = i+1;
                }
                if (vertex_x1_x > vertex_x2_x)
                {
                    i = i - 1;      //(29)   
                    voxel_i = voxel_i-1;
                    next_alpha_index = i-1;                   
                }
                alpha_x = (volumn_x * next_alpha_index + boundary_voxel_x - vertex_x1_x )* inv_x_diff;
                
                if (vertex_x1_y < vertex_x2_y)
                {
					j = j + 1;
                    voxel_j = voxel_j+1;
                    next_alpha_index = j+1;
                }
                else if (vertex_x1_y > vertex_x2_y)
                {
					j = j - 1;
                    voxel_j = voxel_j-1;
                    next_alpha_index = j-1;
                }   
                alpha_y = (volumn_y * next_alpha_index + boundary_voxel_y - vertex_x1_y )* inv_y_diff;
                
                if (vertex_x1_z < vertex_x2_z)
                {
					k = k + 1;
                    voxel_k = voxel_k+1;
                    next_alpha_index = k+1;
                }
                else if (vertex_x1_z > vertex_x2_z)
                {
					k = k - 1;
                    voxel_k = voxel_k-1;
                    next_alpha_index = k-1;
                }  
                alpha_z = (volumn_z * next_alpha_index + BOUNDARY_VOXEL_Z - vertex_x1_z )* inv_z_diff;
           	}
          }
       	}// end tracing the ray                      
     }//else if the ray interacts with the volume
   }//else if the ray is oblique
   
	d_proj_weightedLen[proj_pixel_index] = one_ray_sum ;
    d_proj_sumLen[proj_pixel_index] = one_ray_length;         
    
//    __syncthreads();
    
}




__global__ void backprj_ray_driven_3d_kernel_multiGPU(float *d_volumn_kernel, float *d_proj_correction, float beta_temp, float sin_theta, float cos_theta, int subVolIdx, int command)
{    
    /* 
     * Reference: "Accelerating simultaneous algebraic reconstruction technique with motion compensation using CUDA-enabled GPU" 
     * Wai-Man Pang, CUHK
     * Section: Back-projection and image update
     
     * d_proj_correction : 2D projection correction, i.e. c(i) in the Wai-Man Pang, CUHK paper
     * t_theta : projection angle
     * beta_temp : lamda in the paper
     * d_volumn: 3D object array
     * d_volumn(j) = d_volumn(j) + beta_temp * sum_i (c(i)*w(ij)) / sum_i (w(ij));  where i is ray index, j is voxel index
     */    
       
    int Idx_voxel_x = threadIdx.x + blockIdx.x * blockDim.x;
    int Idx_voxel_y = blockIdx.y;
    int Idx_voxel_z = blockIdx.z;
    
    int image_voxel_index = M * N * Idx_voxel_z + M * Idx_voxel_y + Idx_voxel_x;    

    //coordinate of center of each voxel in x-y-z system     
	float coord_voxel_x = boundary_voxel_x + volumn_x*0.5f + Idx_voxel_x * volumn_x;
    float coord_voxel_y = boundary_voxel_y + volumn_y*0.5f + Idx_voxel_y * volumn_y;
    float coord_voxel_z = boundary_voxel_z + volumn_z*(ZETA/Number_of_Devices*subVolIdx+0.5f) + Idx_voxel_z * volumn_z;   

    /**************************************/
        
	float coord_vertex_x=0.0f, coord_vertex_y=0.0f, coord_vertex_z=0.0f;  
	float coord_vertex_s=0.0f, coord_vertex_t=0.0f;
	float coord_vertexOnDetector_x=0.0f, coord_vertexOnDetector_z=0.0f; 

	float minY = MAX_infi, minZ=MAX_infi, maxY=-MAX_infi, maxZ=-MAX_infi;

	float coord_pixelOnDetector_x=0.0f, coord_pixelOnDetector_y=0.0f, coord_pixelOnDetector_z=0.0f;
	float coord_source_x=0.0f, coord_source_y=0.0f, coord_source_z=0.0f;
	float alpha_x_i_1=0.0f, alpha_x_i=0.0f;
	float alpha_y_i_1=0.0f, alpha_y_i=0.0f;
	float alpha_z_i_1=0.0f, alpha_z_i=0.0f;
   	float alpha_x_temp=0.0f, alpha_y_temp=0.0f, alpha_z_temp=0.0f; 
	float alpha_min=0.0f, alpha_max=0.0f; 
	
	int minY_index=0, maxY_index=0, minZ_index=0, maxZ_index=0; 
	float sumWeight=0.0f, sumLength=0.0f;
	float d_x1_x2=0.0f; 
    float inv_Detector_pixel = 1.0f/Detector_pixel_x;
    	
//     float weight = 1.0f;
//     float tao;
//     float tao_m1 = atan( (float(R)*Detector_pixel_x/2.0f-abs(Offset)) / DSO); 
    

    /***********************************************************/
    
	if ( (Idx_voxel_x-(float(M)*0.5f-0.5)-M_Offset)*volumn_x*(Idx_voxel_x-(float(M)*0.5f-0.5)-M_Offset)*volumn_x 
            +  (Idx_voxel_y-(float(N)*0.5f-0.5))*volumn_y*(Idx_voxel_y-(float(N)*0.5f-0.5))*volumn_y 
            >= (float(M)*0.5f-0.5)*volumn_x*(float(N)*0.5f-0.5)*volumn_y )
    {    
        sumLength = 0.0f;
        sumWeight = 0.0f;
    }
                            
	else
            // Note: The following codes apply to all the voxels simutaneously
	{
        
        /******** investigate the eight vertices of each voxel ********/
        
        for (int k=0;k<2;k++)
            for (int j=0;j<2;j++)
                for (int i=0;i<2;i++)
		{

			//coordinate for each of eight vertices of the voxel 
			coord_vertex_x = coord_voxel_x + (i)*volumn_x - 0.5f*volumn_x; 
			coord_vertex_y = coord_voxel_y + (j)*volumn_y - 0.5f*volumn_y; 
			coord_vertex_z = coord_voxel_z + (k)*volumn_z - 0.5f*volumn_z; 
            
			// <t-s> <----> <x,y>
			coord_vertex_t = coord_vertex_x * cos_theta + coord_vertex_y * sin_theta; 
			coord_vertex_s = - coord_vertex_x * sin_theta + coord_vertex_y * cos_theta;			
            // Note: Now rotate the image volume (with - t_theata degree) instead of the normal gantry rotation
            // In the new coordiantor, detector plane remains and is prependicular to the t axis 
			
            
            // the projcetion of the vertex of the voxel on the detector, in <t,s> system                        
            if (CT_style==0)   //CBCT geometry
            {
                coord_vertexOnDetector_x = (coord_vertex_t - DOD) / (DSO- coord_vertex_t) * (coord_vertex_s - Source_y) + coord_vertex_s ; 
                coord_vertexOnDetector_z = (coord_vertex_t - DOD) / (DSO- coord_vertex_t) * (coord_vertex_z - Source_z) + coord_vertex_z ; 
            }
            else if (CT_style==1)  //FBCT geometry, no magnification along z axis
            {
                coord_vertexOnDetector_x = (coord_vertex_t - DOD) / (DSO- coord_vertex_t) * (coord_vertex_s - Source_y) + coord_vertex_s ; 
                coord_vertexOnDetector_z = coord_voxel_z ;
            }
            else if (CT_style==2)  //PBCT, direct projection
            {
                coord_vertexOnDetector_x = coord_vertex_s;
                coord_vertexOnDetector_z = coord_voxel_z ;
            }
            
            // the projcetion of the vertex of the voxel

			minY= fmin(minY, coord_vertexOnDetector_x);
			maxY= fmax(maxY, coord_vertexOnDetector_x); 
			minZ= fmin(minZ, coord_vertexOnDetector_z);
			maxZ= fmax(maxZ, coord_vertexOnDetector_z); 
            // form a minimim bounding rectangle (MBR) for these vertexes
            
		}

        minY_index = floor( (minY -  Detector_Ymin ) * inv_Detector_pixel +0.5f);
        maxY_index = floor( (maxY -  Detector_Ymin ) * inv_Detector_pixel +0.5f);
        minZ_index = floor( (minZ -  Detector_Zmin ) * inv_Detector_pixel +0.5f);
        maxZ_index = floor( (maxZ -  Detector_Zmin ) * inv_Detector_pixel +0.5f);
        // index of pixels of MBR boudaries on the detector 
               
        /***********************************/

        // If this voxel does not project on this detector plane, it means there is no ray passing throught this voxel at this angle.
        if ((minY_index<0) && (maxY_index <0) || minY_index>(R-1) && maxY_index >(R-1) || (minZ_index<0) && (maxZ_index <0) || (minZ_index>(Z_prj-1)) && (maxZ_index >(Z_prj -1))) 
        {	
            sumWeight = 0.0f;
            sumLength = 0.0f;
        }
                
        else            
            // If this voxel projects on the detector plane 
        {
            
	    	if (minY_index <=0)
        	        minY_index = 0;
        	if (maxY_index >=(R-1) )
                	maxY_index = R-1;
        	if (minZ_index <=0)
                	minZ_index = 0;
        	if (maxZ_index >=(Z_prj-1) )
                	maxZ_index = Z_prj-1;
            
            
            // coordinate of the source  in (x,y,z) system after normal gantry rotation            
            if (CT_style==0)            // CBCT geometry, single source
            {
                coord_source_x = Source_x * cos_theta - Source_y * sin_theta;
                coord_source_y = Source_x * sin_theta + Source_y * cos_theta;
                coord_source_z = Source_z;
            }
            else if (CT_style==1)       // FBCT geometry, multiple sources
            {
                coord_source_x = Source_x * cos_theta - Source_y * sin_theta;
                coord_source_y = Source_x * sin_theta + Source_y * cos_theta;
                coord_source_z = coord_voxel_z;    
            }
            else if (CT_style==2)
            {
                // NOT defined here.
                // The source position goes with the detector element
            }
                                    
            // for those projection pixels whose coordinate loacates inside MBR
            // Each pixel coorresponds to a ray, and that ray must pass through the specific voxel
            for (int j=minZ_index; j<=maxZ_index; j++) 
                for (int i=minY_index; i<=maxY_index; i++)
            {
                coord_pixelOnDetector_x = DOD * cos_theta - (Detector_Ymin + i*Detector_pixel_x) * sin_theta ;
                coord_pixelOnDetector_y = DOD * sin_theta + (Detector_Ymin + i*Detector_pixel_x) * cos_theta ;
                coord_pixelOnDetector_z = Detector_Zmin + j*Detector_pixel_x;                
                // coordinate of the detector pixel inside MBR in (x,y,z) system after normal gantry rotation                   
                
                if (CT_style==2)
                {
                    coord_source_x = Source_x * cos_theta - (Detector_Ymin + i*Detector_pixel_x) * sin_theta;
                    coord_source_y = Source_x * sin_theta + (Detector_Ymin + i*Detector_pixel_x) * cos_theta;
                    coord_source_z = coord_voxel_z;                        
                }
                
                
                /** Weighted Update for Half Detector **/
//                 if ( (float(i)*Detector_pixel_x) < 2.0f*abs(Offset) )
//                     weight = 1.0f;                
//                 else
//                 {
//                     tao = atan( ( float(R/2-i)*Detector_pixel_x + abs(Offset) ) / DSO);                    
//                     weight = cos(PI/4*(tao/tao_m1 - 1));
//                     weight = weight * weight;                     
//                 }
                /******/  
                
                
                // Next: investigate the line starting at x1 and ending at x2 
                                        
                	
                alpha_x_i_1 =  ( (coord_voxel_x - 0.5f*volumn_x) - coord_pixelOnDetector_x )/( coord_source_x - coord_pixelOnDetector_x ); 
                alpha_x_i   =  ( (coord_voxel_x + 0.5f*volumn_x) - coord_pixelOnDetector_x )/( coord_source_x - coord_pixelOnDetector_x ); 
                alpha_y_i_1 =  ( (coord_voxel_y - 0.5f*volumn_y) - coord_pixelOnDetector_y )/( coord_source_y - coord_pixelOnDetector_y ); 
                alpha_y_i   =  ( (coord_voxel_y + 0.5f*volumn_y) - coord_pixelOnDetector_y )/( coord_source_y - coord_pixelOnDetector_y );
                alpha_z_i_1 =  ( (coord_voxel_z - 0.5f*volumn_z) - coord_pixelOnDetector_z )/( coord_source_z - coord_pixelOnDetector_z ); 
                alpha_z_i   =  ( (coord_voxel_z + 0.5f*volumn_z) - coord_pixelOnDetector_z )/( coord_source_z - coord_pixelOnDetector_z ); 
                    // find out indices of the two most closet x planes near this specific voxel

                alpha_x_temp = fmin((alpha_x_i_1), (alpha_x_i));
                alpha_y_temp = fmin((alpha_y_i_1), (alpha_y_i)); 
                alpha_z_temp = fmin((alpha_z_i_1), (alpha_z_i)); 
				alpha_min = fmax(fmax(alpha_x_temp, alpha_y_temp), fmax(alpha_y_temp, alpha_z_temp)); 
                    // alpha_min is the enter point for one specific voxel

                alpha_x_temp = fmax((alpha_x_i_1), (alpha_x_i));
                alpha_y_temp = fmax((alpha_y_i_1), (alpha_y_i));
                alpha_z_temp = fmax((alpha_z_i_1), (alpha_z_i));
				alpha_max = fmin(fmin(alpha_x_temp, alpha_y_temp), fmin(alpha_y_temp, alpha_z_temp));
                    // alpha_max is the exit point of the line passing through this voxel

                if (alpha_max-alpha_min>0)        // if the value is negative, it means the ray does not pass through this voxel
                {
                	d_x1_x2 = sqrt((coord_source_x-coord_pixelOnDetector_x)*(coord_source_x-coord_pixelOnDetector_x) + (coord_source_y-coord_pixelOnDetector_y)*(coord_source_y - coord_pixelOnDetector_y) + (coord_source_z-coord_pixelOnDetector_z)*(coord_source_z-coord_pixelOnDetector_z) );
                    float temp = d_x1_x2*(alpha_max-alpha_min);
                                
                    if  ( temp > volumn_x*1e-6)
                            // the line passes through the voxel with a sufficient length; 
                    {                        
                    	sumWeight  = sumWeight +  temp*d_proj_correction[j*R  + i];
                            // Note: d_proj_correction[j*R + i] is c(i) which has been previously calculated
                            // Note: d_x1_x2 * (alpha_max - alpha_min) is w(i) for ray i of this projection 
                        sumLength = sumLength +  temp;	                        
                    }
                }
			
                
            }// end for loop: all the rays whose projection fits in the rectangle
        }//end else if this voxel projects on this detector plane 
    }//end else if the reconstruction region is in the circle           
    
	if (sumLength < volumn_x*1e-6)
    	d_volumn_kernel[image_voxel_index] += 0.0f ;  
	else
    {                
        if (command==0)
        	d_volumn_kernel[image_voxel_index] = sumWeight ;   // matched ajoint operator, for test use             
        else if (command==1)
        	d_volumn_kernel[image_voxel_index] += beta_temp * sumWeight/sumLength ;                        
	}    
    
//     __syncthreads();
    
}
 