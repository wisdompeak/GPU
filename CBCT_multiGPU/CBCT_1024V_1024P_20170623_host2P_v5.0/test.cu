
#include <hip/hip_runtime.h>
#include <cmath>
#include <fstream>
#include <time.h>
#include <iostream>
#include <stddef.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>

__global__
 void AplusB(int *ret, int a, int b) {
    ret[threadIdx.x] = a + b + threadIdx.x;
}
int main() {
int *ret;
    hipMallocManaged(&ret, 1000 * sizeof(int));
    AplusB<<< 1, 1000 >>>(ret, 10, 100);
        hipDeviceSynchronize();
for(int i=0; i<1000; i++)
        printf("%d: A+B = %d\n", i, ret[i]);
    hipFree(ret); 
return  0;
}