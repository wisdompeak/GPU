void host_forward(int i, float* d_recon, float * d_proj)
{
    dim3  dimblock(256,1);
    dim3  dimgrid(1024,1024/2,1);    
           
    hipSetDevice(i);        
    kernel_forward_projection<<<dimgrid, dimblock>>>(d_recon, d_proj);    
}

void host_backprj(int i, float* d_recon, float * d_proj)
{
    dim3  dimblock(256,1);
    dim3  dimgrid(1024,1024/2,1);    
           
    hipSetDevice(i);        
    kernel_back_projection<<<dimgrid, dimblock>>>(d_recon, d_proj);    
}

void AddProjection(float* d_proj_addr[])
{
    size_t single_gpu_chunk_size = sizeof(float)*Num_elem/2;
    
    dim3  dimblock(256,1);
    dim3  dimgrid(1024,1024/2,1);
    
    hipSetDevice(0);                   
    for (int i=1; i<2; i++)
    {
        hipDeviceEnablePeerAccess(i,0);
        kernel_add<<<dimgrid, dimblock>>>(d_proj_addr[0],d_proj_addr[i]);
    }
    
    for (int i=1; i<2; i++)
    {
        hipMemcpyAsync(d_proj_addr[i],d_proj_addr[0],single_gpu_chunk_size,hipMemcpyDefault);
    }
    
    for (int i=1; i<2; i++)
    {
        hipSetDevice(i);
        hipStreamSynchronize(0);
    }            
}